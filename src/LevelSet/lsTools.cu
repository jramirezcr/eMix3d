#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<math.h>
#include"LevelSet/lsTools.h"
#define MAX(x, y) (((x) > (y)) ? (x) : (y))
#define MIN(x, y) (((x) < (y)) ? (x) : (y))

#define PI 3.14159265359

__device__ double Phi_x_WENO(
                          double beta1,
                          double beta2,
                          double beta3,
                          double beta4,
                          double beta5
                          )
{
   
   double  s_b1, s_b2, s_b3,
          alpha_1, alpha_2, alpha_3,
          omega_1, omega_2, omega_3, result;

   s_b1 = (13.0/12.0)*(beta1 - 2.0*beta2 + beta3)
                     *(beta1 - 2.0*beta2 + beta3)
        + (0.25)*(beta1 - 4.0*beta2 + 3.0*beta3)
                *(beta1 - 4.0*beta2 + 3.0*beta3);

   s_b2 = (13.0/12.0)*(beta2 - 2.0*beta3 + beta4)
                     *(beta2 - 2.0*beta3 + beta4)
        + (0.25)*(beta2 - beta4)*(beta2 - beta4);

   s_b3 = (13.0/12.0)*(beta3 - 2.0*beta4 + beta5)
                     *(beta3 - 2.0*beta4 + beta5)
        + (0.25)*(3.0*beta3 - 4.0*beta4 + beta5)
                *(3.0*beta3 - 4.0*beta4 + beta5);


   alpha_1 = 0.1 /((s_b1 + 1.0e-6)*(s_b1 + 1.0e-6));
   alpha_2 = 0.6 /((s_b2 + 1.0e-6)*(s_b2 + 1.0e-6));
   alpha_3 = 0.3 /((s_b3 + 1.0e-6)*(s_b3 + 1.0e-6));

   omega_1 = alpha_1 / (alpha_1 + alpha_2 + alpha_3);
   omega_2 = alpha_2 / (alpha_1 + alpha_2 + alpha_3);
   omega_3 = alpha_3 / (alpha_1 + alpha_2 + alpha_3);
  
   result = ((omega_1*(2.0*beta1 - 7.0*beta2 + 11.0*beta3) 
       + omega_2*(-1.0*beta2 + 5.0*beta3 + 2.0*beta4)
       + omega_3*(2.0*beta3 + 5.0*beta4 - beta5))*(1.0/6.0));

   return result;
}


__global__ void Dev1thO_Downwind(
                                double* const       d_Phi,
                                const double* const phiS,
                                const double        deltaX,
                                const double        deltaY,
                                const double        deltaZ,
                                const unsigned int  Nx, 
                                const unsigned int  Ny, 
                                const unsigned int  Nz
                                )
{

   const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x,
                      idy = blockIdx.y*blockDim.y + threadIdx.y,
                      idz = blockIdx.z*blockDim.z + threadIdx.z;  

   //Offsets sample (id_ip) EQ (i+1,j,k) 
   unsigned int id = Nx*Ny*idz + Nx*idy + idx,
                id_im = Nx*Ny*idz + Nx*idy + idx - 1, 
                id_jm = Nx*Ny*idz + Nx*(idy - 1) + idx, 
                id_km = Nx*Ny*(idz - 1) + Nx*idy + idx; 
                   
   unsigned int ix = id, 
                iy = id, 
                iz = id;


   //Dealing with boundaries
   if(idx==0){id_im = id; ix = Nx*Ny*idz + Nx*idy + 1;} 
   if(idy==0){id_jm = id; iy = Nx*Ny*idz + Nx*1 + idx;} 
   if(idz==0){id_km = id; iz = Nx*Ny*1 + Nx*idy + idx;} 

   const unsigned int Offset = Nx*Ny*Nz;

   d_Phi[           id] = deltaX*(phiS[ix] - phiS[id_im]);

   d_Phi[1*Offset + id] = deltaY*(phiS[iy] - phiS[id_jm]);

   d_Phi[2*Offset + id] = deltaZ*(phiS[iz] - phiS[id_km]);
	
   return;

}

__global__ void PhiDevPlusParameter(
                                    double* const       phi_xyz,
                                    const double* const d_Phi,
                                    unsigned const int  Nx,
                                    unsigned const int  Ny,
                                    unsigned const int  Nz
                                    )
{
   unsigned const int Offset = Nx*Ny*Nz; 

   unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x,
                idy = blockDim.y*blockIdx.y + threadIdx.y, 
                idz = blockDim.z*blockIdx.z + threadIdx.z;

   unsigned int id = idx + idy*Nx + idz*Nx*Ny,
                id_im1 = (idx - 1) + idy*Nx + idz*Nx*Ny,
                id_ip1 = (idx + 1) + idy*Nx + idz*Nx*Ny,

                id_jm1 = idx + (idy - 1)*Nx + idz*Nx*Ny,
                id_jp1 = idx + (idy + 1)*Nx + idz*Nx*Ny,

                id_km1 = idx + idy*Nx + (idz - 1)*Nx*Ny,
                id_kp1 = idx + idy*Nx + (idz + 1)*Nx*Ny,

                id_im2 = (idx - 2) + idy*Nx + idz*Nx*Ny,
                id_ip2 = (idx + 2) + idy*Nx + idz*Nx*Ny,

                id_jm2 = idx + (idy - 2)*Nx + idz*Nx*Ny,
                id_jp2 = idx + (idy + 2)*Nx + idz*Nx*Ny,

                id_km2 = idx + idy*Nx + (idz - 2)*Nx*Ny,
                id_kp2 = idx + idy*Nx + (idz + 2)*Nx*Ny;


   //Dealing with boundaries

   if(idx == 0    ){id_im1 = id; id_im2 = id_ip1;} 
   if(idx == 1    ){id_im2 = id_im1;} 
   if(idx == Nx -1){id_ip1 = id; id_ip2 = id_im1;} 
   if(idx == Nx -2){id_ip2 = id_ip1;} 

   if(idy == 0    ){id_jm1 = id; id_jm2 = id_jp1;}
   if(idy == 1    ){id_jm2 = id_jm1;} 
   if(idy == Ny -1){id_jp1 = id; id_jp2 = id_jm1;} 
   if(idy == Ny -2){id_jp2 = id_jp1;} 

   if(idz == 0    ){id_km1 = id; id_km2 = id_kp1;} 
   if(idz == 1    ){id_km2 = id_km1;} 
   if(idz == Nz -1){id_kp1 = id; id_kp2 = id_jm1;} 
   if(idz == Nz -2){id_kp2 = id_kp1;} 

   double beta1, beta2, beta3, beta4, beta5;

  
   //Axis X

   beta1 = d_Phi[id_im2];
   beta2 = d_Phi[id_im1]; 
   beta3 = d_Phi[id];
   beta4 = d_Phi[id_ip1];
   beta5 = d_Phi[id_ip2];

   phi_xyz[id] = Phi_x_WENO(beta1, beta2, beta3, beta4, beta5);
   //Axis Y

   beta1 = d_Phi[id_jm2 + 1*Offset];
   beta2 = d_Phi[id_jm1 + 1*Offset]; 
   beta3 = d_Phi[id     + 1*Offset];
   beta4 = d_Phi[id_jp1 + 1*Offset];
   beta5 = d_Phi[id_jp2 + 1*Offset];


   phi_xyz[id + 1*Offset] = Phi_x_WENO(beta1, beta2, beta3, beta4, beta5);

 
   //Axis Z

   beta1 = d_Phi[id_km2 + 2*Offset];
   beta2 = d_Phi[id_km1 + 2*Offset]; 
   beta3 = d_Phi[id     + 2*Offset];
   beta4 = d_Phi[id_kp1 + 2*Offset];
   beta5 = d_Phi[id_kp2 + 2*Offset];

   phi_xyz[id + 2*Offset] = Phi_x_WENO(beta1, beta2, beta3, beta4, beta5);
   return;
}


__global__ void PhiDevMinusParameter(
                                     double* const       phi_xyz,
                                     const double* const d_Phi,
                                     unsigned const int  Nx,
                                     unsigned const int  Ny,
                                     unsigned const int  Nz
                                    )
{
   unsigned const int Offset = Nx*Ny*Nz; 

   unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x,
                idy = blockDim.y*blockIdx.y + threadIdx.y, 
                idz = blockDim.z*blockIdx.z + threadIdx.z;

   unsigned int id = idx + idy*Nx + idz*Nx*Ny,
                id_im1 = (idx - 1) + idy*Nx + idz*Nx*Ny,
                id_im2 = (idx - 2) + idy*Nx + idz*Nx*Ny,
                id_ip1 = (idx + 1) + idy*Nx + idz*Nx*Ny,

                id_jm1 = idx + (idy - 1)*Nx + idz*Nx*Ny,
                id_jm2 = idx + (idy - 2)*Nx + idz*Nx*Ny,
                id_jp1 = idx + (idy + 1)*Nx + idz*Nx*Ny,

                id_km1 = idx + idy*Nx + (idz - 1)*Nx*Ny,
                id_km2 = idx + idy*Nx + (idz - 2)*Nx*Ny,
                id_kp1 = idx + idy*Nx + (idz + 1)*Nx*Ny,

                id_ip2 = (idx + 2) + idy*Nx + idz*Nx*Ny,

                id_jp2 = idx + (idy + 2)*Nx + idz*Nx*Ny,

                id_kp2 = idx + idy*Nx + (idz + 2)*Nx*Ny,

                id_ip3 = (idx + 3) + idy*Nx + idz*Nx*Ny,

                id_jp3 = idx + (idy + 3)*Nx + idz*Nx*Ny,

                id_kp3 = idx + idy*Nx + (idz + 3)*Nx*Ny;

   //Dealing with boundaries

   if(idx == 0    ){id_im1 = id;} 
   if(idx == Nx -1){id_ip1 = id; id_ip2 = id_im1; id_ip3 = id_im2;} 
   if(idx == Nx -2){id_ip2 = id_ip1; id_ip3 = id;} 
   if(idx == Nx -3){id_ip3 = id_ip2;} 


   if(idy == 0    ){id_jm1 = id;} 
   if(idy == Ny -1){id_jp1 = id; id_jp2 = id_jm1; id_jp3 = id_jm2;} 
   if(idy == Ny -2){id_jp2 = id_jp1; id_jp3 = id;} 
   if(idy == Ny -3){id_jp3 = id_jp2;} 

   if(idz == 0    ){id_km1 = id;} 
   if(idz == Nz -1){id_kp1 = id; id_kp2 = id_km1;id_kp3 = id_km2;} 
   if(idz == Nz -2){id_kp2 = id_kp1; id_kp3 = id;} 
   if(idz == Nz -3){id_kp3 = id_kp2;} 


   double beta1, beta2, beta3, beta4, beta5;
  
   //Axis X

   beta1 = d_Phi[id_ip3];
   beta2 = d_Phi[id_ip2]; 
   beta3 = d_Phi[id_ip1];
   beta4 = d_Phi[id    ];
   beta5 = d_Phi[id_im1];


   phi_xyz[id           ] = Phi_x_WENO(beta1, beta2, beta3, beta4, beta5);
   //Axis Y

   beta1 = d_Phi[id_jp3 + 1*Offset];
   beta2 = d_Phi[id_jp2 + 1*Offset]; 
   beta3 = d_Phi[id_jp1 + 1*Offset];
   beta4 = d_Phi[id     + 1*Offset];
   beta5 = d_Phi[id_jm1 + 1*Offset];

   phi_xyz[id + 1*Offset] = Phi_x_WENO(beta1, beta2, beta3, beta4, beta5);

   //Axis Z


   beta1 = d_Phi[id_kp3 + 2*Offset];
   beta2 = d_Phi[id_kp2 + 2*Offset]; 
   beta3 = d_Phi[id_kp1 + 2*Offset];
   beta4 = d_Phi[id     + 2*Offset];
   beta5 = d_Phi[id_km1 + 2*Offset];

   phi_xyz[id + 2*Offset] = Phi_x_WENO(beta1, beta2, beta3, beta4, beta5);
   return;

}


__global__ void reini_RS_WENO(
                             double* const       rs,
                             const double* const phiS,                    
                             const double        deltaXYZ,
                             const double* const d_phiP,
                             const double* const d_phiM,
                             const double* const phiS0,                    
                             unsigned int        Nx,
                             unsigned int        Ny,
                             unsigned int        Nz 
                             )
{

   unsigned int Offset = Nx*Ny*Nz;

   unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x,
                idy = blockDim.y*blockIdx.y + threadIdx.y,
                idz = blockDim.z*blockIdx.z + threadIdx.z;

   unsigned int id = idx + Nx*idy + Nx*Ny*idz;
   double       so, rs_x, rs_y, rs_z, ta, grad_mod;
   double       phiMax, phiMin;

   ta = (double)(phiS0[id] > 0.0) - (double)(phiS0[id] < 0.0);

   //Getting gradient axis X
   phiMax = MAX(d_phiP[id   ], 0.0)*MAX(d_phiP[id   ], 0.0);  
   phiMin = MIN(d_phiM[id   ], 0.0)*MIN(d_phiM[id   ], 0.0);  

   rs_x   = 0.5*(ta + 1.0)*MAX(phiMax, phiMin);

   phiMax = MAX(d_phiM[id   ], 0.0)*MAX(d_phiM[id   ], 0.0);  
   phiMin = MIN(d_phiP[id   ], 0.0)*MIN(d_phiP[id   ], 0.0);  

   rs_x   += 0.5*abs(ta - 1.0)*MAX(phiMax, phiMin);

   //Getting gradient axis Y
   phiMax = MAX(d_phiP[id + 1*Offset], 0.0)
           *MAX(d_phiP[id + 1*Offset], 0.0);  

   phiMin = MIN(d_phiM[id + 1*Offset], 0.0)
           *MIN(d_phiM[id + 1*Offset], 0.0);  

   rs_y   = 0.5*(ta + 1.0)*MAX(phiMax, phiMin);

   phiMax = MAX(d_phiM[id + 1*Offset], 0.0)
           *MAX(d_phiM[id + 1*Offset], 0.0);  

   phiMin = MIN(d_phiP[id + 1*Offset], 0.0)
           *MIN(d_phiP[id + 1*Offset], 0.0);  

   rs_y   += 0.5*abs(ta - 1.0)*MAX(phiMax, phiMin);

   //Getting gradient axis Z
   phiMax = MAX(d_phiP[id + 2*Offset], 0.0)
           *MAX(d_phiP[id + 2*Offset], 0.0);  

   phiMin = MIN(d_phiM[id + 2*Offset], 0.0)
           *MIN(d_phiM[id + 2*Offset], 0.0);  

   rs_z   = 0.5*(ta + 1.0)*MAX(phiMax, phiMin);

   phiMax = MAX(d_phiM[id + 2*Offset], 0.0)
           *MAX(d_phiM[id + 2*Offset], 0.0);  

   phiMin = MIN(d_phiP[id + 2*Offset], 0.0)
           *MIN(d_phiP[id + 2*Offset], 0.0);  

   rs_z   += 0.5*abs(ta - 1.0)*MAX(phiMax, phiMin);

   grad_mod = sqrt(rs_x + rs_y + rs_z);

   so = phiS0[id] 
      / sqrt(phiS0[id]*phiS0[id] + deltaXYZ*deltaXYZ );

   rs[id] = 1.0*so*(grad_mod - 1.0);

   return;
}


__global__ void advect_RS_WENO(
                              double* const       rs,         //RHS 
                              const double* const velocity,   
                              const double* const d_phiP_d,
                              const double* const d_phiM_d,
                              unsigned int        Nx,
                              unsigned int        Ny,
                              unsigned int        Nz
                              )
{
   unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x,
                idy = blockDim.y*blockIdx.y + threadIdx.y,
                idz = blockDim.z*blockIdx.z + threadIdx.z;

   unsigned int id = idx + Nx*idy + Nx*Ny*idz,
                Offset = Nx*Ny*Nz;

   double rs_x, rs_y, rs_z;
   double grad_x, grad_y, grad_z;
   double rsign;

   rsign  = (double)(velocity[id] > 0.0) 
          - (double)(velocity[id] < 0.0); 
            
   rs_x   = 0.5*   (rsign + 1.0)*velocity[id]*d_phiP_d[id] 
          + 0.5*abs(rsign - 1.0)*velocity[id]*d_phiM_d[id];

   grad_x = 0.5*   (rsign + 1.0)*d_phiP_d[id] 
          + 0.5*abs(rsign - 1.0)*d_phiM_d[id];

   rsign  = (double)(velocity[id + 1*Offset] > 0.0) 
          - (double)(velocity[id + 1*Offset] < 0.0); 

   rs_y   = 0.5*(rsign + 1.0)*velocity[id + 1*Offset]
               *d_phiP_d[id + 1*Offset] 
          + 0.5*abs(rsign - 1.0)*velocity[id + 1*Offset]
               *d_phiM_d[id + 1*Offset];

   grad_y = 0.5*   (rsign + 1.0)*d_phiP_d[id + 1*Offset] 
          + 0.5*abs(rsign - 1.0)*d_phiM_d[id + 1*Offset];

   rsign  = (double)(velocity[id + 2*Offset] > 0.0) 
          - (double)(velocity[id + 2*Offset] < 0.0); 

   rs_z   = 0.5*(rsign + 1.0)*velocity[id + 2*Offset]
               *d_phiP_d[id + 2*Offset] 
          + 0.5*abs(rsign - 1.0)*velocity[id + 2*Offset]
               *d_phiM_d[id + 2*Offset];

   grad_z = 0.5*   (rsign + 1.0)*d_phiP_d[id + 2*Offset] 
          + 0.5*abs(rsign - 1.0)*d_phiM_d[id + 2*Offset];


   rs[id] = rs_x + rs_y + rs_z;
   
   return;
}


__global__
void enrightVelocityProfile(
                            double       *vel,    //Velocity Array
                            double       *xMesh,  //Mesh values 
                            double       *yMesh,
                            double       *zMesh,
                            const int     Nx,     //Mesh dimensions
                            const int     Ny,
                            const int     Nz,
                            const double  time,   //current time
                            const double  period 
                           )
{

   unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x,
                idy = blockDim.y*blockIdx.y + threadIdx.y,
                idz = blockDim.z*blockIdx.z + threadIdx.z;

   unsigned int id = idx + Nx*idy + Nx*Ny*idz,
                offset = Nx*Ny*Nz;
   
   vel[id           ] = 2.0*sin(PI*xMesh[id])*sin(PI*xMesh[id])
                           *sin(2.0*PI*yMesh[id])
                           *sin(2.0*PI*zMesh[id])*cos(PI*time/period);

   vel[id + 1*offset] =   -sin(PI*yMesh[id])*sin(PI*yMesh[id])
                          *sin(2.0*PI*xMesh[id])
                          *sin(2.0*PI*zMesh[id])*cos(PI*time/period);

   vel[id + 2*offset] =   -sin(PI*zMesh[id])*sin(PI*zMesh[id])
                          *sin(2.0*PI*yMesh[id])
                          *sin(2.0*PI*xMesh[id])*cos(PI*time/period);

      
}

__global__
void meshRegularStructured(
                           double       *xMesh,  //Mesh values 
                           double       *yMesh,
                           double       *zMesh,
                           double        deltaX,
                           double        deltaY,
                           double        deltaZ,
                           const int     Nx,     //Mesh dimensions
                           const int     Ny,
                           const int     Nz
                          )
{

   unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x,
                idy = blockDim.y*blockIdx.y + threadIdx.y,
                idz = blockDim.z*blockIdx.z + threadIdx.z;

   unsigned int id   = idx + Nx*idy + Nx*Ny*idz;

   xMesh[id] = (double)(idx - 5.0)*deltaX;   
   yMesh[id] = (double)(idy - 5.0)*deltaY;   
   zMesh[id] = (double)(idz - 5.0)*deltaZ;   

}

__global__
void cuGhostCellsMirror3dZ( 
                          double    *ghostArray,
                          const int  ncells,
                          const int  Nx,
                          const int  Ny,
                          const int  Nz,
                          double     direction
                          )
{

   int NxG = Nx + 2*ncells,
       NyG = Ny + 2*ncells;

   unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x,
                idy = blockDim.y*blockIdx.y + threadIdx.y;

   //Left boundary
   for(unsigned int idz = 0; idz < ncells; idz++){
      unsigned int id  = idx + NxG*idy + NxG*NyG*(ncells + idz);
      unsigned int idG = idx + NxG*idy + NxG*NyG*(ncells - idz - 1);

      ghostArray[idG] = ghostArray[id]*direction;    
   }

   //right boundary
   for(unsigned int idz = 0; idz < ncells; idz++){
      unsigned int id  = idx + NxG*idy + NxG*NyG*(Nz - idz - 1 + ncells);
      unsigned int idG = idx + NxG*idy + NxG*NyG*(Nz + idz + ncells);

      ghostArray[idG] = ghostArray[id]*direction;    
   }
}

__global__
void cuGhostCellsMirror3dY( 
                          double    *ghostArray,
                          const int  ncells,
                          const int  Nx,
                          const int  Ny,
                          const int  Nz,
                          double     direction
                          )
{

   int NxG = Nx + 2*ncells,
       NyG = Ny + 2*ncells,
       NzG = Nz + 2*ncells;

   unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x,
                idz = blockDim.y*blockIdx.y + threadIdx.y;

   //Left boundary
   for(unsigned int idy = 0; idy < ncells; idy++){
      unsigned int id  = idx + NxG*(ncells + idy) + NxG*NyG*idz;
      unsigned int idG = idx + NxG*(ncells - idy - 1) + NxG*NyG*idz;

      ghostArray[idG] = ghostArray[id]*direction;    
   }

   //right boundary
   for(unsigned int idy = 0; idy < ncells; idy++){
      unsigned int id  = idx + NxG*(Ny - idy - 1 + ncells) + NxG*NyG*idz;
      unsigned int idG = idx + NxG*(Ny + idy + ncells) + NxG*NyG*idz;

      ghostArray[idG] = ghostArray[id]*direction;    
   }
}

__global__
void cuGhostCellsMirror3dX( 
                          double    *ghostArray,
                          const int  ncells,
                          const int  Nx,
                          const int  Ny,
                          const int  Nz,
                          double     direction
                          )
{


   int NxG = Nx + 2*ncells,
       NyG = Ny + 2*ncells,
       NzG = Nz + 2*ncells;

   unsigned int idy = blockDim.x*blockIdx.x + threadIdx.x,
                idz = blockDim.y*blockIdx.y + threadIdx.y;

   //Left boundary
   for(unsigned int idx = 0; idx < ncells; idx++){
      unsigned int id  = (ncells + idx) + NxG*idy + NxG*NyG*idz;
      unsigned int idG = (ncells - idx - 1) + NxG*idy + NxG*NyG*idz;

      ghostArray[idG] = ghostArray[id]*direction;    
   }

   //right boundary
   for(unsigned int idx = 0; idx < ncells; idx++){
      unsigned int id  = (Nx - idx - 1 + ncells) + NxG*idy + NxG*NyG*idz;
      unsigned int idG = (Nx + idx + ncells) + NxG*idy + NxG*NyG*idz;

      ghostArray[idG] = ghostArray[id]*direction;    
   }
}

__global__ void PhiDevPlusParameterJB(
                                    double* const       phi_xyz,
                                    const double* const d_Phi,
                                    const double* const d_jbn,
                                    unsigned const int  Nx,
                                    unsigned const int  Ny,
                                    unsigned const int  Nz
                                    )
{
   unsigned const int Offset = Nx*Ny*Nz; 

   unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x,
                idy = blockDim.y*blockIdx.y + threadIdx.y, 
                idz = blockDim.z*blockIdx.z + threadIdx.z;

   unsigned int id = idx + idy*Nx + idz*Nx*Ny,
                id_im1 = (idx - 1) + idy*Nx + idz*Nx*Ny,
                id_ip1 = (idx + 1) + idy*Nx + idz*Nx*Ny,

                id_jm1 = idx + (idy - 1)*Nx + idz*Nx*Ny,
                id_jp1 = idx + (idy + 1)*Nx + idz*Nx*Ny,

                id_km1 = idx + idy*Nx + (idz - 1)*Nx*Ny,
                id_kp1 = idx + idy*Nx + (idz + 1)*Nx*Ny,

                id_im2 = (idx - 2) + idy*Nx + idz*Nx*Ny,
                id_ip2 = (idx + 2) + idy*Nx + idz*Nx*Ny,

                id_jm2 = idx + (idy - 2)*Nx + idz*Nx*Ny,
                id_jp2 = idx + (idy + 2)*Nx + idz*Nx*Ny,

                id_km2 = idx + idy*Nx + (idz - 2)*Nx*Ny,
                id_kp2 = idx + idy*Nx + (idz + 2)*Nx*Ny;


   //Dealing with boundaries

   if(idx == 0    ){id_im1 = id; id_im2 = id_ip1;} 
   if(idx == 1    ){id_im2 = id_im1;} 
   if(idx == Nx -1){id_ip1 = id; id_ip2 = id_im1;} 
   if(idx == Nx -2){id_ip2 = id_ip1;} 

   if(idy == 0    ){id_jm1 = id; id_jm2 = id_jp1;}
   if(idy == 1    ){id_jm2 = id_jm1;} 
   if(idy == Ny -1){id_jp1 = id; id_jp2 = id_jm1;} 
   if(idy == Ny -2){id_jp2 = id_jp1;} 

   if(idz == 0    ){id_km1 = id; id_km2 = id_kp1;} 
   if(idz == 1    ){id_km2 = id_km1;} 
   if(idz == Nz -1){id_kp1 = id; id_kp2 = id_jm1;} 
   if(idz == Nz -2){id_kp2 = id_kp1;} 

   double beta1, beta2, beta3, beta4, beta5;

  
   //Axis X

   beta1 = d_Phi[id_im2]*d_jbn[id];
   beta2 = d_Phi[id_im1]*d_jbn[id]; 
   beta3 = d_Phi[id    ]*d_jbn[id];
   beta4 = d_Phi[id_ip1]*d_jbn[id];
   beta5 = d_Phi[id_ip2]*d_jbn[id];

   phi_xyz[id] = Phi_x_WENO(beta1, beta2, beta3, beta4, beta5);
   //Axis Y

   beta1 = d_Phi[id_jm2 + 1*Offset]*d_jbn[id + 4*Offset];
   beta2 = d_Phi[id_jm1 + 1*Offset]*d_jbn[id + 4*Offset]; 
   beta3 = d_Phi[id     + 1*Offset]*d_jbn[id + 4*Offset];
   beta4 = d_Phi[id_jp1 + 1*Offset]*d_jbn[id + 4*Offset];
   beta5 = d_Phi[id_jp2 + 1*Offset]*d_jbn[id + 4*Offset];


   phi_xyz[id + 1*Offset] = Phi_x_WENO(beta1, beta2, beta3, beta4, beta5);

 
   //Axis Z

   beta1 = d_Phi[id_km2 + 2*Offset]*d_jbn[id + 8*Offset];
   beta2 = d_Phi[id_km1 + 2*Offset]*d_jbn[id + 8*Offset]; 
   beta3 = d_Phi[id     + 2*Offset]*d_jbn[id + 8*Offset];
   beta4 = d_Phi[id_kp1 + 2*Offset]*d_jbn[id + 8*Offset];
   beta5 = d_Phi[id_kp2 + 2*Offset]*d_jbn[id + 8*Offset];

   phi_xyz[id + 2*Offset] = Phi_x_WENO(beta1, beta2, beta3, beta4, beta5);
   return;
}


__global__ void PhiDevMinusParameterJB(
                                     double* const       phi_xyz,
                                     const double* const d_Phi,
                                     const double* const d_jbn,
                                     unsigned const int  Nx,
                                     unsigned const int  Ny,
                                     unsigned const int  Nz
                                    )
{
   unsigned const int Offset = Nx*Ny*Nz; 

   unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x,
                idy = blockDim.y*blockIdx.y + threadIdx.y, 
                idz = blockDim.z*blockIdx.z + threadIdx.z;

   unsigned int id = idx + idy*Nx + idz*Nx*Ny,
                id_im1 = (idx - 1) + idy*Nx + idz*Nx*Ny,
                id_im2 = (idx - 2) + idy*Nx + idz*Nx*Ny,
                id_ip1 = (idx + 1) + idy*Nx + idz*Nx*Ny,

                id_jm1 = idx + (idy - 1)*Nx + idz*Nx*Ny,
                id_jm2 = idx + (idy - 2)*Nx + idz*Nx*Ny,
                id_jp1 = idx + (idy + 1)*Nx + idz*Nx*Ny,

                id_km1 = idx + idy*Nx + (idz - 1)*Nx*Ny,
                id_km2 = idx + idy*Nx + (idz - 2)*Nx*Ny,
                id_kp1 = idx + idy*Nx + (idz + 1)*Nx*Ny,

                id_ip2 = (idx + 2) + idy*Nx + idz*Nx*Ny,

                id_jp2 = idx + (idy + 2)*Nx + idz*Nx*Ny,

                id_kp2 = idx + idy*Nx + (idz + 2)*Nx*Ny,

                id_ip3 = (idx + 3) + idy*Nx + idz*Nx*Ny,

                id_jp3 = idx + (idy + 3)*Nx + idz*Nx*Ny,

                id_kp3 = idx + idy*Nx + (idz + 3)*Nx*Ny;

   //Dealing with boundaries

   if(idx == 0    ){id_im1 = id;} 
   if(idx == Nx -1){id_ip1 = id; id_ip2 = id_im1; id_ip3 = id_im2;} 
   if(idx == Nx -2){id_ip2 = id_ip1; id_ip3 = id;} 
   if(idx == Nx -3){id_ip3 = id_ip2;} 


   if(idy == 0    ){id_jm1 = id;} 
   if(idy == Ny -1){id_jp1 = id; id_jp2 = id_jm1; id_jp3 = id_jm2;} 
   if(idy == Ny -2){id_jp2 = id_jp1; id_jp3 = id;} 
   if(idy == Ny -3){id_jp3 = id_jp2;} 

   if(idz == 0    ){id_km1 = id;} 
   if(idz == Nz -1){id_kp1 = id; id_kp2 = id_km1;id_kp3 = id_km2;} 
   if(idz == Nz -2){id_kp2 = id_kp1; id_kp3 = id;} 
   if(idz == Nz -3){id_kp3 = id_kp2;} 


   double beta1, beta2, beta3, beta4, beta5;
  
   //Axis X

   beta1 = d_Phi[id_ip3]*d_jbn[id];
   beta2 = d_Phi[id_ip2]*d_jbn[id]; 
   beta3 = d_Phi[id_ip1]*d_jbn[id];
   beta4 = d_Phi[id    ]*d_jbn[id];
   beta5 = d_Phi[id_im1]*d_jbn[id];


   phi_xyz[id           ] = Phi_x_WENO(beta1, beta2, beta3, beta4, beta5);
   //Axis Y

   beta1 = d_Phi[id_jp3 + 1*Offset]*d_jbn[id + 4*Offset];
   beta2 = d_Phi[id_jp2 + 1*Offset]*d_jbn[id + 4*Offset]; 
   beta3 = d_Phi[id_jp1 + 1*Offset]*d_jbn[id + 4*Offset];
   beta4 = d_Phi[id     + 1*Offset]*d_jbn[id + 4*Offset];
   beta5 = d_Phi[id_jm1 + 1*Offset]*d_jbn[id + 4*Offset];

   phi_xyz[id + 1*Offset] = Phi_x_WENO(beta1, beta2, beta3, beta4, beta5);

   //Axis Z


   beta1 = d_Phi[id_kp3 + 2*Offset]*d_jbn[id + 8*Offset];
   beta2 = d_Phi[id_kp2 + 2*Offset]*d_jbn[id + 8*Offset]; 
   beta3 = d_Phi[id_kp1 + 2*Offset]*d_jbn[id + 8*Offset];
   beta4 = d_Phi[id     + 2*Offset]*d_jbn[id + 8*Offset];
   beta5 = d_Phi[id_km1 + 2*Offset]*d_jbn[id + 8*Offset];

   phi_xyz[id + 2*Offset] = Phi_x_WENO(beta1, beta2, beta3, beta4, beta5);

   return;

}

__global__
void cuSwapToGhost( 
                   double    *ghostArray,
                   double    *valueArray,
                   const int  gcells,
                   const int  Nx,
                   const int  Ny,
                   const int  Nz
                  )
{

   int NxG = Nx + 2*gcells; 
   int NyG = Ny + 2*gcells; 

   unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x,
                idy = blockDim.y*blockIdx.y + threadIdx.y, 
                idz = blockDim.z*blockIdx.z + threadIdx.z;

   unsigned int id  = idx + idy*Nx + idz*Nx*Ny;
   unsigned int idG = idx + gcells + (idy + gcells)*NxG 
                    + (idz + gcells)*NxG*NyG;

   ghostArray[idG] = valueArray[id];

}

__global__
void cuSwapFromGhost( 
                   double    *valueArray,
                   double    *ghostArray,
                   const int  gcells,
                   const int  Nx,
                   const int  Ny,
                   const int  Nz
                  )
{

   int NxG = Nx + 2*gcells; 
   int NyG = Ny + 2*gcells; 

   unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x,
                idy = blockDim.y*blockIdx.y + threadIdx.y, 
                idz = blockDim.z*blockIdx.z + threadIdx.z;

   unsigned int id  = idx + idy*Nx + idz*Nx*Ny;
   unsigned int idG = idx + gcells + (idy + gcells)*NxG 
                    + (idz + gcells)*NxG*NyG;

   valueArray[id] = ghostArray[idG];

}

__global__ void reini_RS_WENOJB(
                             double* const       rs,
                             const double* const phiS,                    
                             const double* const deltaXYZ,
                             const double* const d_phiP,
                             const double* const d_phiM,
                             const double* const phiS0,                    
                             unsigned int        Nx,
                             unsigned int        Ny,
                             unsigned int        Nz 
                             )
{

   unsigned int Offset = Nx*Ny*Nz;

   unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x,
                idy = blockDim.y*blockIdx.y + threadIdx.y,
                idz = blockDim.z*blockIdx.z + threadIdx.z;

   unsigned int id = idx + Nx*idy + Nx*Ny*idz;
   double       so, rs_x, rs_y, rs_z, ta, grad_mod;
   double       phiMax, phiMin;

   ta = (double)(phiS0[id] > 0.0) - (double)(phiS0[id] < 0.0);

   //Getting gradient axis X
   phiMax = MAX(d_phiP[id   ], 0.0)*MAX(d_phiP[id   ], 0.0);  
   phiMin = MIN(d_phiM[id   ], 0.0)*MIN(d_phiM[id   ], 0.0);  

   rs_x   = 0.5*(ta + 1.0)*MAX(phiMax, phiMin);

   phiMax = MAX(d_phiM[id   ], 0.0)*MAX(d_phiM[id   ], 0.0);  
   phiMin = MIN(d_phiP[id   ], 0.0)*MIN(d_phiP[id   ], 0.0);  

   rs_x   += 0.5*abs(ta - 1.0)*MAX(phiMax, phiMin);

   //Getting gradient axis Y
   phiMax = MAX(d_phiP[id + 1*Offset], 0.0)
           *MAX(d_phiP[id + 1*Offset], 0.0);  

   phiMin = MIN(d_phiM[id + 1*Offset], 0.0)
           *MIN(d_phiM[id + 1*Offset], 0.0);  

   rs_y   = 0.5*(ta + 1.0)*MAX(phiMax, phiMin);

   phiMax = MAX(d_phiM[id + 1*Offset], 0.0)
           *MAX(d_phiM[id + 1*Offset], 0.0);  

   phiMin = MIN(d_phiP[id + 1*Offset], 0.0)
           *MIN(d_phiP[id + 1*Offset], 0.0);  

   rs_y   += 0.5*abs(ta - 1.0)*MAX(phiMax, phiMin);

   //Getting gradient axis Z
   phiMax = MAX(d_phiP[id + 2*Offset], 0.0)
           *MAX(d_phiP[id + 2*Offset], 0.0);  

   phiMin = MIN(d_phiM[id + 2*Offset], 0.0)
           *MIN(d_phiM[id + 2*Offset], 0.0);  

   rs_z   = 0.5*(ta + 1.0)*MAX(phiMax, phiMin);

   phiMax = MAX(d_phiM[id + 2*Offset], 0.0)
           *MAX(d_phiM[id + 2*Offset], 0.0);  

   phiMin = MIN(d_phiP[id + 2*Offset], 0.0)
           *MIN(d_phiP[id + 2*Offset], 0.0);  

   rs_z   += 0.5*abs(ta - 1.0)*MAX(phiMax, phiMin);

   grad_mod = sqrt(rs_x + rs_y + rs_z);

   so = phiS0[id] 
      / sqrt(phiS0[id]*phiS0[id] + deltaXYZ[id]*deltaXYZ[id] );

   rs[id] = 1.0*so*(grad_mod - 1.0);

   return;
}

