#include<hip/hip_runtime.h>
//only kernels

__device__ double devDiv(
                         const double c1,
                         const double e1,
                         const double c2,
                         const double e2,
                         const double c3,
                         const double e3,
                         const double delta
                         )
{ 
   return delta*(c1*e1+c2*e2+c3*e3);
}


__global__ void divDevXPlus(
                           double* const d_func,
                           const double* const func,
                           const unsigned int Nx, 	
                           const unsigned int Ny, 	
                           const unsigned int Nz,  	
                           const double deltax,
                           const double c1,
                           const double c2,
                           const double c3
                           ) 
{

   double e1, e2, e3;
   unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x,
                idy = blockDim.y*blockIdx.y + threadIdx.y,
                idz = blockDim.z*blockIdx.z + threadIdx.z;


   unsigned int id = idx + Nx*idy + Nx*Ny*idz,
                id_ip1 = (idx + 1) + Nx*idy + Nx*Ny*idz,
                id_ip2 = (idx + 2) + Nx*idy + Nx*Ny*idz;

  
   e1 = func[id];
   e2 = func[id_ip1];
   e3 = func[id_ip2];

   if(idx == Nx-2){

      unsigned int id_ipNxm1 = (Nx - 1) + Nx*idy + Nx*Ny*idz,
                   id_ipNxm2 = (Nx - 2) + Nx*idy + Nx*Ny*idz,
                   id_ipNxm3 = (Nx - 3) + Nx*idy + Nx*Ny*idz,
                   id_ipNxm4 = (Nx - 4) + Nx*idy + Nx*Ny*idz;

      e1 = func[id_ipNxm2];
      e2 = func[id_ipNxm1];
      e3 = 4.0*func[id_ipNxm1] - 6.0*func[id_ipNxm2] + 4.0*func[id_ipNxm3] 
         - func[id_ipNxm4];
   }

   if(idx == Nx-1){
      e1 = 0.0;
      e2 = 0.0;
      e3 = 0.0;
   }


   d_func[id] = devDiv(c1, e1, c2, e2, c3, e3, deltax);

   return;
}

__global__ void divDevXMin(
                           double* const d_func,
                           const double* const func,
                           const unsigned int Nx, 	
                           const unsigned int Ny, 	
                           const unsigned int Nz,  	
                           const double deltax,
                           const double c1,
                           const double c2,
                           const double c3
                           ) 
{

   double e1, e2, e3;
   unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x,
                idy = blockDim.y*blockIdx.y + threadIdx.y,
                idz = blockDim.z*blockIdx.z + threadIdx.z;


   unsigned int id = idx + Nx*idy + Nx*Ny*idz,
                id_im1 = (idx - 1) + Nx*idy + Nx*Ny*idz,
                id_im2 = (idx - 2) + Nx*idy + Nx*Ny*idz;
  
   e1 = func[id];
   e2 = func[id_im1];
   e3 = func[id_im2];

   if(idx == 1){

   unsigned int id_im0 = Nx*idy + Nx*Ny*idz,
                id_im0p1 = 1 + Nx*idy + Nx*Ny*idz,
                id_im0p2 = 2 + Nx*idy + Nx*Ny*idz,
                id_im0p3 = 3 + Nx*idy + Nx*Ny*idz;

      e1 = func[id_im0p1];
      e2 = func[id_im0];
      e3 = 4.0*func[id_im0] - 6.0*func[id_im0p1] + 4.0*func[id_im0p2] 
         - func[id_im0p3];
   }

   if(idx == 0){
      e1 = 0.0;
      e2 = 0.0;
      e3 = 0.0;
   }


   d_func[id] = devDiv(c1, e1, c2, e2, c3, e3, deltax);

   return;
}



__global__ void divDevYPlus(
                           double* const d_func,
                           const double* const func,
                           const unsigned int Nx, 	
                           const unsigned int Ny, 	
                           const unsigned int Nz,  	
                           const double deltaY,
                           const double c1,
                           const double c2,
                           const double c3
                           ) 
{

   double e1, e2, e3;
   unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x,
                idy = blockDim.y*blockIdx.y + threadIdx.y,
                idz = blockDim.z*blockIdx.z + threadIdx.z;


   unsigned int id = idx + Nx*idy + Nx*Ny*idz,
                id_jp1 = idx + Nx*(idy + 1) + Nx*Ny*idz,
                id_jp2 = idx + Nx*(idy + 2) + Nx*Ny*idz;

  
   e1 = func[id];
   e2 = func[id_jp1];
   e3 = func[id_jp2];

   if(idy == Ny-2){

      unsigned int id_jpNym1 = idx + Nx*(Ny -1) + Nx*Ny*idz,
                   id_jpNym2 = idx + Nx*(Ny -2) + Nx*Ny*idz,
                   id_jpNym3 = idx + Nx*(Ny -3) + Nx*Ny*idz,
                   id_jpNym4 = idx + Nx*(Ny -4) + Nx*Ny*idz;

      e1 = func[id_jpNym2];
      e2 = func[id_jpNym1];
      e3 = 4.0*func[id_jpNym1] - 6.0*func[id_jpNym2] + 4.0*func[id_jpNym3] 
         - func[id_jpNym4];
   }

   if(idy == Ny - 1){
      e1 = 0.0;
      e2 = 0.0;
      e3 = 0.0;
   }


   d_func[id] = devDiv(c1, e1, c2, e2, c3, e3, deltaY);

   return;


}

__global__ void divDevYMin(
                           double* const d_func,
                           const double* const func,
                           const unsigned int Nx, 	
                           const unsigned int Ny, 	
                           const unsigned int Nz,  	
                           const double deltaY,
                           const double c1,
                           const double c2,
                           const double c3
                           ) 
{

   double e1, e2, e3;
   unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x,
                idy = blockDim.y*blockIdx.y + threadIdx.y,
                idz = blockDim.z*blockIdx.z + threadIdx.z;


   unsigned int id = idx + Nx*idy + Nx*Ny*idz,
                id_jm1 = idx + Nx*(idy - 1) + Nx*Ny*idz,
                id_jm2 = idx + Nx*(idy - 2) + Nx*Ny*idz;
  
   e1 = func[id];
   e2 = func[id_jm1];
   e3 = func[id_jm2];

   if(idy == 1){

   unsigned int id_jm0 = idx + Nx*Ny*idz,
                id_jm0p1 = idx + Nx*1 + Nx*Ny*idz,
                id_jm0p2 = idx + Nx*2 + Nx*Ny*idz,
                id_jm0p3 = idx + Nx*3 + Nx*Ny*idz;

      e1 = func[id_jm0p1];
      e2 = func[id_jm0];
      e3 = 4.0*func[id_jm0] - 6.0*func[id_jm0p1] + 4.0*func[id_jm0p2] 
         - func[id_jm0p3];
   }

   if(idy == 0){
      e1 = 0.0;
      e2 = 0.0;
      e3 = 0.0;
   }


   d_func[id] = devDiv(c1, e1, c2, e2, c3, e3, deltaY);

   return;
}


__global__ void divDevZPlus(
                           double* const d_func,
                           const double* const func,
                           const unsigned int Nx, 	
                           const unsigned int Ny, 	
                           const unsigned int Nz,  	
                           const double deltaZ,
                           const double c1,
                           const double c2,
                           const double c3
                           ) 
{

   double e1, e2, e3;
   unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x,
                idy = blockDim.y*blockIdx.y + threadIdx.y,
                idz = blockDim.z*blockIdx.z + threadIdx.z;


   unsigned int id = idx + Nx*idy + Nx*Ny*idz,
                id_kp1 = idx + Nx*idy + Nx*Ny*(idz + 1),
                id_kp2 = idx + Nx*idy + Nx*Ny*(idz + 2);

  
   e1 = func[id];
   e2 = func[id_kp1];
   e3 = func[id_kp2];

   if(idz == Nz - 2){

      unsigned int id_kpNzm1 = idx + Nx*idy + Nx*Ny*(Nz - 1),
                   id_kpNzm2 = idx + Nx*idy + Nx*Ny*(Nz - 2),
                   id_kpNzm3 = idx + Nx*idy + Nx*Ny*(Nz - 3),
                   id_kpNzm4 = idx + Nx*idy + Nx*Ny*(Nz - 4);

      e1 = func[id_kpNzm2];
      e2 = func[id_kpNzm1];
      e3 = 4.0*func[id_kpNzm1] - 6.0*func[id_kpNzm2] + 4.0*func[id_kpNzm3] 
         - func[id_kpNzm4];
   }

   if(idz == Nz - 1){
      e1 = 0.0;
      e2 = 0.0;
      e3 = 0.0;
   }


   d_func[id] = devDiv(c1, e1, c2, e2, c3, e3, deltaZ);

   return;
}

__global__ void divDevZMin(
                           double* const d_func,
                           const double* const func,
                           const unsigned int Nx, 	
                           const unsigned int Ny, 	
                           const unsigned int Nz,  	
                           const double deltaZ,
                           const double c1,
                           const double c2,
                           const double c3
                           ) 
{

   double e1, e2, e3;
   unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x,
                idy = blockDim.y*blockIdx.y + threadIdx.y,
                idz = blockDim.z*blockIdx.z + threadIdx.z;


   unsigned int id = idx + Nx*idy + Nx*Ny*idz,
                id_km1 = idx + Nx*idy + Nx*Ny*(idz - 1),
                id_km2 = idx + Nx*idy + Nx*Ny*(idz - 2);
  
   e1 = func[id];
   e2 = func[id_km1];
   e3 = func[id_km2];

   if(idz == 1){

   unsigned int id_km0 = idx + Nx*Ny*idz,
                id_km0p1 = idx + idy + Nx*Ny*1,
                id_km0p2 = idx + idy + Nx*Ny*2,
                id_km0p3 = idx + idy + Nx*Ny*3;

      e1 = func[id_km0p1];
      e2 = func[id_km0];
      e3 = 4.0*func[id_km0] - 6.0*func[id_km0p1] + 4.0*func[id_km0p2] 
         - func[id_km0p3];
   }

   if(idz == 0){
      e1 = 0.0;
      e2 = 0.0;
      e3 = 0.0;
   }


   d_func[id] = devDiv(c1, e1, c2, e2, c3, e3, deltaZ);

   return;
}




__global__ void get_flux_e_CUDA(
                               double* const flux,
                               const double* const e_f,
                               const double* const f_f,
                               const double* const g_f,
                               const double* const jbn,
                               const unsigned int Nx, 
                               const unsigned int Ny, 
                               const unsigned int Nz 
                               )
{
   unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x,
                idy = blockDim.y*blockIdx.y + threadIdx.y,
                idz = blockDim.z*blockIdx.z + threadIdx.z;

   unsigned int id = idx + Nx*idy + Nx*Ny*idz, 
                Offset = Nx*Ny*Nz;

   flux[id] = jbn[id + 9*Offset]*(
                                 jbn[id + 0*Offset]*e_f[id] + 
                                 jbn[id + 1*Offset]*f_f[id] + 
                                 jbn[id + 2*Offset]*g_f[id] 
                                 );


   return;
}

__global__ void get_flux_f_CUDA(
                               double* const flux,
                               const double* const e_f,
                               const double* const f_f,
                               const double* const g_f,
                               const double* const jbn,
                               const unsigned int Nx, 
                               const unsigned int Ny, 
                               const unsigned int Nz 
                               )
{
   unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x,
                idy = blockDim.y*blockIdx.y + threadIdx.y,
                idz = blockDim.z*blockIdx.z + threadIdx.z;

   unsigned int id = idx + Nx*idy + Nx*Ny*idz, 
                Offset = Nx*Ny*Nz;

   flux[id] = jbn[id + 9*Offset]*(
                                 jbn[id + 3*Offset]*e_f[id] + 
                                 jbn[id + 4*Offset]*f_f[id] + 
                                 jbn[id + 5*Offset]*g_f[id] 
                                 );


   return;
}

__global__ void get_flux_g_CUDA(
                               double* const flux,
                               const double* const e_f,
                               const double* const f_f,
                               const double* const g_f,
                               const double* const jbn,
                               const unsigned int Nx, 
                               const unsigned int Ny, 
                               const unsigned int Nz 
                               )
{
   unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x,
                idy = blockDim.y*blockIdx.y + threadIdx.y,
                idz = blockDim.z*blockIdx.z + threadIdx.z;

   unsigned int id = idx + Nx*idy + Nx*Ny*idz, 
                Offset = Nx*Ny*Nz;

   flux[id] = jbn[id + 9*Offset]*(
                                 jbn[id + 6*Offset]*e_f[id] + 
                                 jbn[id + 7*Offset]*f_f[id] + 
                                 jbn[id + 8*Offset]*g_f[id]  
                                 );

   return;
}

__global__ void rs_divergence_CUDA(
                                   double* const rs,
                                   const double* const e_x,
                                   const double* const f_y,
                                   const double* const g_z,
                                   const double* const jbn,
                                   unsigned int Nx, 
                                   unsigned int Ny, 
                                   unsigned int Nz
                                   )
{
   unsigned int idx = threadIdx.x + blockIdx.x*blockDim.x,
                idy = threadIdx.y + blockIdx.y*blockDim.y,
                idz = threadIdx.z + blockIdx.z*blockDim.z;
 
   unsigned int id = idx + Nx*idy + Nx*Ny*idz,
                Offset = Nx*Ny*Nz;

   rs[id] = jbn[id + 10*Offset]*(
                                 e_x[id] + e_x[id + Offset] 
                               + f_y[id] + f_y[id + Offset] 
                               + g_z[id] + g_z[id + Offset] 
                                );

   return;
}
