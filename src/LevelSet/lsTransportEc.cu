#include<hip/hip_runtime.h>
#include<iostream>
#include <fstream>
#include<stdio.h>
#include<stdlib.h>
#include"LevelSet/lsTools.h"
#include"LevelSet/extrapol.h"
#include"LevelSet/lsTransportEc.h"


extern "C"{
/*
    dim3 dimBlock(5,5,5);
    dim3 dimGrid(32,32,32);

    dim3 dimBlockG(5,5,5);
    dim3 dimGridG(30,30,30);

    dim3 dimBlockB(5,5);
    dim3 dimGridB(32,32);
*/


void advect_CUDA(
                 double *press,
                 double *jbn,
                 double *sobject,
                 double  deltaX,
                 double  deltaY,
                 double  deltaZ,
                 int     Nx, 
                 int     Ny, 
                 int     Nz,
                 double  deltamin
               )
{


   int gcells = 5;

   double *d_jbn;
   double *d_jbnW;
   double *d_sobject;
   double *d_sobjectW;
   double *d_press;
   double *d_pressW;
   

   int NxG = Nx + 2*gcells,
       NyG = Ny + 2*gcells,
       NzG = Nz + 2*gcells; 

   int offset  = Nx*Ny*Nz;
   int offsetG = NxG*NyG*NzG;

    int numGBX, numGBY, numGBZ;

    dim3 dimBlock(10,10,5);

    numGBX = NxG / 10;
    numGBY = NyG / 10;
    numGBZ = NzG / 5;

    dim3 dimGrid(numGBX,numGBY,numGBZ);

    dim3 dimBlockG(10,10,5);

    numGBX = Nx / 10;
    numGBY = Ny / 10;
    numGBZ = Nz / 5;

    dim3 dimGridG(numGBX,numGBY,numGBZ);

    dim3 dimBlockB(10,10);

    numGBX = NxG / 10;
    numGBY = NyG / 10;

    dim3 dimGridB(numGBX,numGBY);

    hipSetDevice(1);

   hipMalloc((void**)&d_jbnW,      11*sizeof(double)*offset);
   hipMalloc((void**)&d_sobjectW,     sizeof(double)*offset);
   hipMalloc((void**)&d_pressW,       sizeof(double)*offset);

   hipMalloc((void**)&d_jbn,     11*sizeof(double)*offsetG);
   hipMalloc((void**)&d_sobject,    sizeof(double)*offsetG);
   hipMalloc((void**)&d_press,      sizeof(double)*offsetG);

   hipMemcpy(d_jbnW,     jbn,    11*sizeof(double)*offset,
              hipMemcpyHostToDevice);
   hipMemcpy(d_sobjectW, sobject,   sizeof(double)*offset,
              hipMemcpyHostToDevice);
   hipMemcpy(d_pressW,press,        sizeof(double)*offset,
              hipMemcpyHostToDevice);



   cuSwapToGhost<<<dimGridG, dimBlockG>>>
                 (d_press, d_pressW, gcells, Nx, Ny,Nz);

   cuSwapToGhost<<<dimGridG, dimBlockG>>>
            (&(d_jbn[0*offsetG]), &(d_jbnW[0*offset]), gcells, Nx, Ny,Nz);

   cuSwapToGhost<<<dimGridG, dimBlockG>>>
            (&(d_jbn[4*offsetG]), &(d_jbnW[4*offset]), gcells, Nx, Ny,Nz);

   cuSwapToGhost<<<dimGridG, dimBlockG>>>
            (&(d_jbn[8*offsetG]), &(d_jbnW[8*offset]), gcells, Nx, Ny,Nz);

   cuSwapToGhost<<<dimGridG, dimBlockG>>>
                 (d_sobject, d_sobjectW, gcells, Nx, Ny,Nz);


   //Boundary Conditions Ghost Cells press

   cuGhostCellsMirror3dZ<<<dimGridB, dimBlockB>>>
                        (d_press, gcells, Nx, Ny, Nz, 1.0);

   cuGhostCellsMirror3dY<<<dimGridB, dimBlockB>>>
                        (d_press, gcells, Nx, Ny, Nz, 1.0);

   cuGhostCellsMirror3dX<<<dimGridB, dimBlockB>>>
                        (d_press, gcells, Nx, Ny, Nz, 1.0);

   //Boundary Conditions Ghost Cells geometry

   cuGhostCellsMirror3dZ<<<dimGridB, dimBlockB>>>
                        (d_sobject, gcells, Nx, Ny, Nz, 1.0);

   cuGhostCellsMirror3dY<<<dimGridB, dimBlockB>>>
                        (d_sobject, gcells, Nx, Ny, Nz, 1.0);

   cuGhostCellsMirror3dX<<<dimGridB, dimBlockB>>>
                        (d_sobject, gcells, Nx, Ny, Nz, 1.0);


   //Ghost cells Jacobean

   cuGhostCellsMirror3dX<<<dimGridB, dimBlockB>>>
                        (&(d_jbn[0*offsetG]), gcells, Nx, Ny, Nz,  1.0);

   cuGhostCellsMirror3dY<<<dimGridB, dimBlockB>>>
                        (&(d_jbn[0*offsetG]), gcells, Nx, Ny, Nz,  1.0);

   cuGhostCellsMirror3dZ<<<dimGridB, dimBlockB>>>
                        (&(d_jbn[0*offsetG]), gcells, Nx, Ny, Nz,  1.0);

   cuGhostCellsMirror3dX<<<dimGridB, dimBlockB>>>
                        (&(d_jbn[4*offsetG]), gcells, Nx, Ny, Nz,  1.0);

   cuGhostCellsMirror3dY<<<dimGridB, dimBlockB>>>
                        (&(d_jbn[4*offsetG]), gcells, Nx, Ny, Nz,  1.0);

   cuGhostCellsMirror3dZ<<<dimGridB, dimBlockB>>>
                        (&(d_jbn[4*offsetG]), gcells, Nx, Ny, Nz,  1.0);

   cuGhostCellsMirror3dX<<<dimGridB, dimBlockB>>>
                        (&(d_jbn[8*offsetG]), gcells, Nx, Ny, Nz,  1.0);

   cuGhostCellsMirror3dY<<<dimGridB, dimBlockB>>>
                        (&(d_jbn[8*offsetG]), gcells, Nx, Ny, Nz,  1.0);

   cuGhostCellsMirror3dZ<<<dimGridB, dimBlockB>>>
                        (&(d_jbn[8*offsetG]), gcells, Nx, Ny, Nz,  1.0);


   //Extrapolating pressure and ls (geometry boundary condition)
   cuExtrapolation(d_press,d_sobject,d_jbn,deltaX,deltaY,deltaZ, 
                   NxG, NyG, NzG, deltamin*0.5, 1.0);

   cuSwapFromGhost<<<dimGridG, dimBlockG>>>
                  (d_pressW, d_press, gcells, Nx, Ny,Nz);

   //Returning values from gpu to cpu (phi, vel, press)
   hipMemcpy(press,d_pressW,sizeof(double)*offset, hipMemcpyDeviceToHost);

   hipFree(d_jbn);
   hipFree(d_jbnW);
   hipFree(d_sobject);
   hipFree(d_sobjectW);
   hipFree(d_press);
   hipFree(d_pressW);

   // check for error
   hipError_t error = hipGetLastError();
   if(error != hipSuccess)
   {
     // print the CUDA error message and exit
     std::cout << "CUDA error extrapol:\n" <<  hipGetErrorString(error) 
               << std::endl;
     exit(-1);
   }


} 

}// end extern C

void cuAdvectLsJB(
                 double *d_phi, 
                 double *d_vel,
                 double *d_jbn,
                 double  deltaX,
                 double  deltaY,
                 double  deltaZ,
                 double  dt,
                 int     NxG, 
                 int     NyG, 
                 int     NzG
                 )
{

    hipSetDevice(1);

    //Derived of Level-Set function
    double *d_dPhi;
    double *d_rsPhi;
    double *d_phiTemp;
    double *d_dPhiPlus;
    double *d_dPhiMinus;

    int offsetG = NxG*NyG*NzG; 

    int numGBX,numGBY,numGBZ;
   
    dim3 dimBlock(10,10,5);

    numGBX = NxG / 10;
    numGBY = NyG / 10;
    numGBZ = NxG / 5;

    dim3 dimGrid(numGBX,numGBY,numGBZ);

    hipMalloc((void**)&d_dPhi,      3*sizeof(double)*offsetG);
    hipMalloc((void**)&d_dPhiPlus,  3*sizeof(double)*offsetG);
    hipMalloc((void**)&d_dPhiMinus, 3*sizeof(double)*offsetG);
    hipMalloc((void**)&d_rsPhi,       sizeof(double)*offsetG);
    hipMalloc((void**)&d_phiTemp,     sizeof(double)*offsetG);


    //Runge First Step
    Dev1thO_Downwind<<<dimGrid,dimBlock>>>(
                                          d_dPhi,
                                          d_phi,
                                          1.0/deltaX,
                                          1.0/deltaY,
                                          1.0/deltaZ,
                                          NxG, 
                                          NyG, 
                                          NzG
                                          );


    PhiDevPlusParameterJB<<<dimGrid,dimBlock>>>(
                                              d_dPhiPlus,
                                              d_dPhi,
                                              d_jbn,
                                              NxG, 
                                              NyG, 
                                              NzG 
                                             );  


    PhiDevMinusParameterJB<<<dimGrid,dimBlock>>>( d_dPhiMinus,
                                                  d_dPhi,
                                                  d_jbn,
                                               NxG, 
                                               NyG, 
                                               NzG 
                                              );  

    advect_RS_WENO<<<dimGrid,dimBlock>>>(
                                         d_rsPhi, 
                                         d_vel,
                                         d_dPhiPlus,
                                         d_dPhiMinus,
                                         NxG, 
                                         NyG, 
                                         NzG
                                        );

     RunGK_FirstS<<<dimGrid,dimBlock>>>(
                                        d_phiTemp,
                                        d_phi, 
                                        dt, 
                                        d_rsPhi, 
                                        NxG, 
                                        NyG, 
                                        NzG
                                       );  

    //Second step Runge-Kutta
    Dev1thO_Downwind<<<dimGrid,dimBlock>>>(
                                          d_dPhi,
                                          d_phiTemp,
                                          1.0/deltaX,
                                          1.0/deltaY,
                                          1.0/deltaZ,
                                          NxG, 
                                          NyG, 
                                          NzG
                                          );


    PhiDevMinusParameterJB<<<dimGrid,dimBlock>>>(
                                               d_dPhiMinus,
                                               d_dPhi,
                                               d_jbn,
                                               NxG, 
                                               NyG, 
                                               NzG 
                                              );  

    PhiDevPlusParameterJB<<<dimGrid,dimBlock>>>(
                                              d_dPhiPlus,
                                              d_dPhi,
                                              d_jbn,
                                              NxG, 
                                              NyG, 
                                              NzG 
                                             );  


    advect_RS_WENO<<<dimGrid,dimBlock>>>(
                                         d_rsPhi, 
                                         d_vel,
                                         d_dPhiPlus,
                                         d_dPhiMinus,
                                         NxG, 
                                         NyG, 
                                         NzG
                                        );


    RunGK_SecondS<<<dimGrid, dimBlock>>>(
                                         d_phiTemp, 
                                         d_phi, 
                                         d_phiTemp, 
                                         dt,
                                         d_rsPhi, 
                                         NxG, 
                                         NyG, 
                                         NzG
                                        );

    //Third step Runge-Kutta
    Dev1thO_Downwind<<<dimGrid,dimBlock>>>(
                                          d_dPhi,
                                          d_phiTemp,
                                          1.0/deltaX,
                                          1.0/deltaY,
                                          1.0/deltaZ,
                                          NxG, 
                                          NyG, 
                                          NzG
                                          );


    PhiDevMinusParameterJB<<<dimGrid,dimBlock>>>(
                                               d_dPhiMinus,
                                               d_dPhi,
                                               d_jbn,
                                               NxG, 
                                               NyG, 
                                               NzG 
                                              );  


    PhiDevPlusParameterJB<<<dimGrid,dimBlock>>>(
                                              d_dPhiPlus,
                                              d_dPhi,
                                              d_jbn,
                                              NxG, 
                                              NyG, 
                                              NzG 
                                             );  

    advect_RS_WENO<<<dimGrid,dimBlock>>>(
                                         d_rsPhi, 
                                         d_vel,
                                         d_dPhiPlus,
                                         d_dPhiMinus,
                                         NxG, 
                                         NyG, 
                                         NzG
                                        );

    RunGK_ThirdS<<<dimGrid, dimBlock>>>(
                                        d_phi, 
                                        d_phi, 
                                        d_phiTemp, 
                                        dt,
                                        d_rsPhi, 
                                        NxG, 
                                        NyG, 
                                        NzG
                                        );


   // check for error
   hipError_t error = hipGetLastError();
   if(error != hipSuccess)
   {
     // print the CUDA error message and exit
     std::cout << "CUDA error cuAdvectLsJB:\n" <<  hipGetErrorString(error) 
               << std::endl;
     exit(-1);
   }

   hipFree(d_dPhi);
   hipFree(d_phiTemp);
   hipFree(d_rsPhi);
   hipFree(d_dPhiPlus);
   hipFree(d_dPhiMinus);

}

     

void cuReinitLsJB(
                 double *d_phi,    // Level set function on DEVICE
                 double *d_jbn,    // Level set function on DEVICE
                 double  deltaX,
                 double  deltaY,
                 double  deltaZ,
                 double *d_dmins,    // Level set function on DEVICE
                 int     Nx, 
                 int     Ny, 
                 int     Nz,
                 int     gcells,
                 double  deltamin
                 )
{


    int numGBX,numGBY,numGBZ;
   
    dim3 dimBlock(10,10,5);

    numGBX = Nx / 10;
    numGBY = Ny / 10;
    numGBZ = Nz / 5;

    dim3 dimGrid(numGBX,numGBY,numGBZ);

    dim3 dimBlockB(10,10);

    numGBX = Nx / 10;
    numGBY = Ny / 10;

    dim3 dimGridB(numGBX,numGBY);

    //Derived of Level-Set function
    double *d_dPhi;
    double *d_phi0;
    double *d_rsPhi;
    double *d_phiTemp;
    double *d_dPhiPlus;
    double *d_dPhiMinus;

    hipMalloc((void**)&d_dPhi,      3*sizeof(double)*Nx*Ny*Nz);
    hipMalloc((void**)&d_phi0,        sizeof(double)*Nx*Ny*Nz);
    hipMalloc((void**)&d_dPhiPlus,  3*sizeof(double)*Nx*Ny*Nz);
    hipMalloc((void**)&d_dPhiMinus, 3*sizeof(double)*Nx*Ny*Nz);
    hipMalloc((void**)&d_rsPhi,       sizeof(double)*Nx*Ny*Nz);
    hipMalloc((void**)&d_phiTemp,     sizeof(double)*Nx*Ny*Nz);
  

    for(int itera = 1; itera <= 10; itera++){

        //Boundary Conditions Ghost Cells Phi

        cuGhostCellsMirror3dZ<<<dimGridB, dimBlockB>>>
        (d_phi, gcells, Nx-2*gcells, Ny-2*gcells, Nz-2*gcells, 1.0);

        cuGhostCellsMirror3dY<<<dimGridB, dimBlockB>>>
        (d_phi, gcells, Nx-2*gcells, Ny-2*gcells, Nz-2*gcells, 1.0);

        cuGhostCellsMirror3dX<<<dimGridB, dimBlockB>>>
        (d_phi, gcells, Nx-2*gcells, Ny-2*gcells, Nz-2*gcells, 1.0);


        hipMemcpy(d_phi0,d_phi,sizeof(double)*Nx*Ny*Nz,
                  hipMemcpyDeviceToDevice );


        //Runge First Step
        Dev1thO_Downwind<<<dimGrid,dimBlock>>>(
                                              d_dPhi,
                                              d_phi,
                                              1.0/deltaX,
                                              1.0/deltaY,
                                              1.0/deltaZ,
                                              Nx, 
                                              Ny, 
                                              Nz
                                              );
       
       
        PhiDevPlusParameterJB<<<dimGrid,dimBlock>>>(
                                                  d_dPhiPlus,
                                                  d_dPhi,
                                                  d_jbn,
                                                  Nx, 
                                                  Ny, 
                                                  Nz 
                                                 );  
       
       
        PhiDevMinusParameterJB<<<dimGrid,dimBlock>>>(
                                                   d_dPhiMinus,
                                                   d_dPhi,
                                                   d_jbn,
                                                   Nx, 
                                                   Ny, 
                                                   Nz 
                                                  );  
       
        reini_RS_WENOJB<<<dimGrid,dimBlock>>>(
                                             d_rsPhi, 
                                             d_phi, 
                                             d_dmins,
                                             d_dPhiPlus,
                                             d_dPhiMinus,
                                             d_phi0, 
                                             Nx, 
                                             Ny, 
                                             Nz
                                            );
       
         RunGK_FirstS<<<dimGrid,dimBlock>>>(
                                            d_phiTemp,
                                            d_phi, 
                                            0.5*deltamin,
                                            d_rsPhi, 
                                            Nx, 
                                            Ny, 
                                            Nz
                                           );  
       
        //Second step Runge-Kutta
        Dev1thO_Downwind<<<dimGrid,dimBlock>>>(
                                              d_dPhi,
                                              d_phiTemp,
                                              1.0/deltaX,
                                              1.0/deltaY,
                                              1.0/deltaZ,
                                              Nx, 
                                              Ny, 
                                              Nz
                                              );
       
       
        PhiDevMinusParameterJB<<<dimGrid,dimBlock>>>(
                                                   d_dPhiMinus,
                                                   d_dPhi,
                                                   d_jbn,
                                                   Nx, 
                                                   Ny, 
                                                   Nz 
                                                  );  
       
        PhiDevPlusParameterJB<<<dimGrid,dimBlock>>>(
                                                  d_dPhiPlus,
                                                  d_dPhi,
                                                  d_jbn,
                                                  Nx, 
                                                  Ny, 
                                                  Nz 
                                                 );  
       
       
        reini_RS_WENOJB<<<dimGrid,dimBlock>>>(
                                             d_rsPhi, 
                                             d_phiTemp,
                                             d_dmins,
                                             d_dPhiPlus,
                                             d_dPhiMinus,
                                             d_phi0, 
                                             Nx, 
                                             Ny, 
                                             Nz
                                            );
       
       
        RunGK_SecondS<<<dimGrid, dimBlock>>>(
                                             d_phiTemp, 
                                             d_phi, 
                                             d_phiTemp, 
                                             0.5*deltamin,
                                             d_rsPhi, 
                                             Nx, 
                                             Ny, 
                                             Nz
                                            );
       
        //Third step Runge-Kutta
        Dev1thO_Downwind<<<dimGrid,dimBlock>>>(
                                              d_dPhi,
                                              d_phiTemp,
                                              1.0/deltaX,
                                              1.0/deltaY,
                                              1.0/deltaZ,
                                              Nx, 
                                              Ny, 
                                              Nz
                                              );
       
       
        PhiDevMinusParameterJB<<<dimGrid,dimBlock>>>(
                                                   d_dPhiMinus,
                                                   d_dPhi,
                                                   d_jbn,
                                                   Nx, 
                                                   Ny, 
                                                   Nz 
                                                  );  
       
       
        PhiDevPlusParameterJB<<<dimGrid,dimBlock>>>(
                                                  d_dPhiPlus,
                                                  d_dPhi,
                                                  d_jbn,
                                                  Nx, 
                                                  Ny, 
                                                  Nz 
                                                 );  

       
        reini_RS_WENOJB<<<dimGrid,dimBlock>>>(
                                             d_rsPhi, 
                                             d_phiTemp,
                                             d_dmins,
                                             d_dPhiPlus,
                                             d_dPhiMinus,
                                             d_phi0, 
                                             Nx, 
                                             Ny, 
                                             Nz
                                            );
       

        RunGK_ThirdS<<<dimGrid, dimBlock>>>(
                                            d_phi, 
                                            d_phi, 
                                            d_phiTemp, 
                                            0.5*deltamin,
                                            d_rsPhi, 
                                            Nx, 
                                            Ny, 
                                            Nz
                                        );

   }

   // check for error
   hipError_t error = hipGetLastError();
   if(error != hipSuccess)
   {
     // print the CUDA error message and exit
     std::cout << "CUDA error cuReinitLsJB: \n" <<  hipGetErrorString(error) 
               << std::endl;
     exit(-1);
   }

   hipFree(d_dPhi);
   hipFree(d_phi0);
   hipFree(d_phiTemp);
   hipFree(d_rsPhi);
   hipFree(d_dPhiPlus);
   hipFree(d_dPhiMinus);
 

}


