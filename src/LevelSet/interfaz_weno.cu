#include<hip/hip_runtime.h>
#include<stdio.h>
#include"extrapol.h"
#include"womegas.h"
#include"dimdef.h"


extern "C"{
void reiniz_CUDA(
                double* const phiS,
                const double* const jbn,
                const double* const deltaXYZ,
                const double deltaX,
                const double deltaY,
                const double deltaZ,
                int Nx,
                int Ny,
                int Nz,
                const double dtin
                )
{
   double *phiS_d, *phiS0_d,*jbn_d, *deltaXYZ_d,
          *d_phiP_d, *d_phiM_d, *rs_d, *d_Phi_d, *d_d;   

   unsigned int Offset = Nx*Ny*Nz;
   hipMalloc((void**)&phiS_d,     sizeof(double)*Offset);
   hipMalloc((void**)&phiS0_d,    sizeof(double)*Offset);
   hipMalloc((void**)&jbn_d,      sizeof(double)*11*Offset);
   hipMalloc((void**)&deltaXYZ_d, sizeof(double)*Offset);
   hipMalloc((void**)&d_Phi_d,    sizeof(double)*3*Offset);
   hipMalloc((void**)&d_phiP_d,   sizeof(double)*3*Offset);
   hipMalloc((void**)&d_phiM_d,   sizeof(double)*3*Offset);
   hipMalloc((void**)&rs_d,       sizeof(double)*Offset);
   hipMalloc((void**)&d_d,        sizeof(double)*Offset);

   hipMemcpy(phiS_d, phiS, sizeof(double)*Offset, 
              hipMemcpyHostToDevice);
   hipMemcpy(phiS0_d, phiS, sizeof(double)*Offset, 
              hipMemcpyHostToDevice);
   hipMemcpy(jbn_d, jbn, sizeof(double)*11*Offset, 
              hipMemcpyHostToDevice);
   hipMemcpy(deltaXYZ_d, deltaXYZ, sizeof(double)*Offset, 
              hipMemcpyHostToDevice);

   dim3 DimBlock(BLOCKDMX,BLOCKDMY,BLOCKDMZ);   
   dim3 DimGrid(GRIDMX,GRIDMY,GRIDMZ);   

   for(int itera = 1 ; itera <= 20; itera++){
       //First Step
       
       Dev1thO_Downwind<<<DimGrid, DimBlock>>>( d_Phi_d, phiS_d,
                                    deltaX, deltaY, deltaZ,
                                    Nx, Ny, Nz);
       PhiDevPlusParameter<<<DimGrid, DimBlock>>>( d_phiP_d, d_Phi_d,
                                                   jbn_d, Nx, Ny, Nz);
     
       PhiDevMinusParameter<<<DimGrid, DimBlock>>>( d_phiM_d, d_Phi_d,
                                                    jbn_d, Nx, Ny, Nz);
     
       reini_RS_WENO<<<DimGrid, DimBlock>>>(rs_d, phiS_d, phiS0_d,deltaXYZ_d,
                                            d_phiP_d, d_phiM_d, Nx, Ny, Nz);

       RunGK_FirstS<<<DimGrid, DimBlock>>>(d_d, phiS_d, dtin, 
                                           rs_d, Nx, Ny, Nz);  

       //Second Step
     
       Dev1thO_Downwind<<<DimGrid, DimBlock>>>( d_Phi_d, d_d,
                                    deltaX, deltaY, deltaZ,
                                    Nx, Ny, Nz);
     
       PhiDevPlusParameter<<<DimGrid, DimBlock>>>( d_phiP_d, d_Phi_d,
                                                   jbn_d, Nx, Ny, Nz);
     
       PhiDevMinusParameter<<<DimGrid, DimBlock>>>( d_phiM_d, d_Phi_d,
                                                    jbn_d, Nx, Ny, Nz);
     
       reini_RS_WENO<<<DimGrid, DimBlock>>>(rs_d, d_d,phiS0_d, deltaXYZ_d,
                                            d_phiP_d, d_phiM_d, Nx, Ny, Nz );
     
       RunGK_SecondS<<<DimGrid, DimBlock>>>(d_d, phiS_d, d_d, dtin,
                                            rs_d, Nx, Ny, Nz);  
     
     
       //Third Step
       Dev1thO_Downwind<<<DimGrid, DimBlock>>>( d_Phi_d, d_d,
                                    deltaX, deltaY, deltaZ,
                                    Nx, Ny, Nz);
     
       PhiDevPlusParameter<<<DimGrid, DimBlock>>>( d_phiP_d, d_Phi_d,
                                                   jbn_d, Nx, Ny, Nz);
     
       PhiDevMinusParameter<<<DimGrid, DimBlock>>>( d_phiM_d, d_Phi_d,
                                                    jbn_d, Nx, Ny, Nz);
     
       reini_RS_WENO<<<DimGrid, DimBlock>>>(rs_d, d_d,phiS0_d, deltaXYZ_d,
                                          d_phiP_d, d_phiM_d, Nx, Ny, Nz );
     
       RunGK_ThirdS<<<DimGrid, DimBlock>>>(phiS_d, phiS_d, d_d, dtin,
                                            rs_d, Nx, Ny, Nz);  
   } 


   hipMemcpy(phiS, phiS_d, sizeof(double)*Offset, 
              hipMemcpyDeviceToHost);

 // check for error
  hipError_t error = hipGetLastError();
  if(error != hipSuccess)
  {
    // print the CUDA error message and exit
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }


   hipFree(phiS_d);
   hipFree(phiS0_d);
   hipFree(jbn_d);
   hipFree(deltaXYZ_d);
   hipFree(d_phiP_d);
   hipFree(d_phiM_d);
   hipFree(d_Phi_d);
   hipFree(rs_d);
   hipFree(d_d);
   return;
}
}

extern "C"{
void advect_CUDA(
                double* const phiS,
                const double* const velocity,
                const double* const jbn,
                double deltaX,
                double deltaY,
                double deltaZ,
                unsigned int Nx, unsigned int Ny, unsigned int Nz,
                const double dt
                )
{
   double *phiS_d, *velocity_d, *jbn_d,
          *rs_d, *d_d, *d_Phi_d, *d_phiP_d, *d_phiM_d;

   unsigned int Offset = Nx*Ny*Nz;

   hipMalloc((void**)&phiS_d, sizeof(double)*Offset);
   hipMalloc((void**)&velocity_d, sizeof(double)*3*Offset);
   hipMalloc((void**)&jbn_d, sizeof(double)*11*Offset);
   hipMalloc((void**)&rs_d, sizeof(double)*Offset);
   hipMalloc((void**)&d_d, sizeof(double)*Offset);
   hipMalloc((void**)&d_Phi_d, sizeof(double)*3*Offset);
   hipMalloc((void**)&d_phiP_d, sizeof(double)*3*Offset);
   hipMalloc((void**)&d_phiM_d, sizeof(double)*3*Offset);

   hipMemcpy(phiS_d, phiS, sizeof(double)*Offset, 
              hipMemcpyHostToDevice);
   hipMemcpy(velocity_d, velocity, sizeof(double)*3*Offset, 
              hipMemcpyHostToDevice);
   hipMemcpy(jbn_d, jbn, sizeof(double)*11*Offset, 
              hipMemcpyHostToDevice);


   dim3 DimBlock(BLOCKDMX,BLOCKDMY,BLOCKDMZ);   
   dim3 DimGrid(GRIDMX,GRIDMY,GRIDMZ);   

   Dev1thO_Downwind<<<DimGrid, DimBlock>>>( d_Phi_d, phiS_d,
                                deltaX, deltaY, deltaZ,
                                Nx, Ny, Nz);
   PhiDevPlusParameter<<<DimGrid, DimBlock>>>( d_phiP_d, d_Phi_d,
                                               jbn_d, Nx, Ny, Nz);
 
   PhiDevMinusParameter<<<DimGrid, DimBlock>>>( d_phiM_d, d_Phi_d,
                                                jbn_d, Nx, Ny, Nz);
 
   advect_RS_WENO<<<DimGrid, DimBlock>>>(rs_d, velocity_d,
                                        d_phiP_d, d_phiM_d, Nx, Ny, Nz);

   RunGK_FirstS<<<DimGrid, DimBlock>>>(d_d, phiS_d, dt, 
                                       rs_d, Nx, Ny, Nz);  

   //Second Step
 
   Dev1thO_Downwind<<<DimGrid, DimBlock>>>( d_Phi_d, d_d,
                                deltaX, deltaY, deltaZ,
                                Nx, Ny, Nz);
 
   PhiDevPlusParameter<<<DimGrid, DimBlock>>>( d_phiP_d, d_Phi_d,
                                               jbn_d, Nx, Ny, Nz);
 
   PhiDevMinusParameter<<<DimGrid, DimBlock>>>( d_phiM_d, d_Phi_d,
                                                jbn_d, Nx, Ny, Nz);
 

   advect_RS_WENO<<<DimGrid, DimBlock>>>(rs_d, velocity_d,
                                        d_phiP_d, d_phiM_d, Nx, Ny, Nz);
 
   RunGK_SecondS<<<DimGrid, DimBlock>>>(d_d, phiS_d, d_d, dt,
                                        rs_d, Nx, Ny, Nz);  
 
 
   //Third Step
   Dev1thO_Downwind<<<DimGrid, DimBlock>>>( d_Phi_d, d_d,
                                deltaX, deltaY, deltaZ,
                                Nx, Ny, Nz);
 
   PhiDevPlusParameter<<<DimGrid, DimBlock>>>( d_phiP_d, d_Phi_d,
                                               jbn_d, Nx, Ny, Nz);
 
   PhiDevMinusParameter<<<DimGrid, DimBlock>>>( d_phiM_d, d_Phi_d,
                                                jbn_d, Nx, Ny, Nz);
 

   advect_RS_WENO<<<DimGrid, DimBlock>>>(rs_d, velocity_d,
                                        d_phiP_d, d_phiM_d, Nx, Ny, Nz);
 
   RunGK_ThirdS<<<DimGrid, DimBlock>>>(phiS_d, phiS_d, d_d, dt,
                                           rs_d, Nx, Ny, Nz);  


   hipMemcpy(phiS, phiS_d, sizeof(double)*Offset, 
              hipMemcpyDeviceToHost);
   
   
   // check for error
   hipError_t error = hipGetLastError();
   if(error != hipSuccess)
   {
     // print the CUDA error message and exit
      printf("CUDA error: %s\n", hipGetErrorString(error));
      exit(-1);
   }
   
   hipFree(phiS_d);
   hipFree(jbn_d);
   hipFree(velocity_d);
   hipFree(d_phiP_d);
   hipFree(d_phiM_d);
   hipFree(d_Phi_d);
   hipFree(rs_d);
   hipFree(d_d);

   return;
}
}

