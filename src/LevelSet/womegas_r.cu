#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<math.h>
#include"womegas.h"
#define MAX(x, y) (((x) > (y)) ? (x) : (y))
#define MIN(x, y) (((x) < (y)) ? (x) : (y))

__device__ double Phi_x_WENO(
                          double beta1,
                          double beta2,
                          double beta3,
                          double beta4,
                          double beta5
                          )
{
   
   double  s_b1, s_b2, s_b3,
          alpha_1, alpha_2, alpha_3,
          omega_1, omega_2, omega_3, result;
   s_b1 = (13.0/12.0)*(beta1 - 2.0*beta2 + beta3)*(beta1 - 2.0*beta2 + beta3)
        + (0.25)*(beta1 - 4.0*beta2 + 3.0*beta3)*(beta1 - 4.0*beta2 + 3.0*beta3);

   s_b2 = (13.0/12.0)*(beta2 - 2.0*beta3 + beta4)*(beta2 - 2.0*beta3 + beta4)
        + (0.25)*(beta2 - beta4)*(beta2 - beta4);

   s_b3 = (13.0/12.0)*(beta3 - 2.0*beta4 + beta5)*(beta3 - 2.0*beta4 + beta5)
        + (0.25)*(3.0*beta3 - 4.0*beta4 + beta5)*(3.0*beta3 - 4.0*beta4 + beta5);


   alpha_1 = 0.1 /((s_b1 + 1.0e-6)*(s_b1 + 1.0e-6));
   alpha_2 = 0.6 /((s_b2 + 1.0e-6)*(s_b2 + 1.0e-6));
   alpha_3 = 0.3 /((s_b3 + 1.0e-6)*(s_b3 + 1.0e-6));

   omega_1 = alpha_1 / (alpha_1 + alpha_2 + alpha_3);
   omega_2 = alpha_2 / (alpha_1 + alpha_2 + alpha_3);
   omega_3 = alpha_3 / (alpha_1 + alpha_2 + alpha_3);
  
   result = ((omega_1*(2.0*beta1 - 7.0*beta2 + 11.0*beta3) 
       + omega_2*(-1.0*beta2 + 5.0*beta3 + 2.0*beta4)
       + omega_3*(2.0*beta3 + 5.0*beta4 - beta5))*(1.0/6.0));

   return result;
}


__global__ void Dev1thO_Downwind(
                                double* const d_Phi,
                                const double* const phiS,
                                const double deltaX,
                                const double deltaY,
                                const double deltaZ,
                                const unsigned int Nx, 
                                const unsigned int Ny, 
                                const unsigned int Nz
                                )
{

   const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x,
                      idy = blockIdx.y*blockDim.y + threadIdx.y,
                      idz = blockIdx.z*blockDim.z + threadIdx.z;  

   //Offsets sample (id_ip) EQ (i+1,j,k) 
   unsigned int id = Nx*Ny*idz + Nx*idy + idx,
                id_im = Nx*Ny*idz + Nx*idy + idx - 1, 
                id_jm = Nx*Ny*idz + Nx*(idy - 1) + idx, 
                id_km = Nx*Ny*(idz - 1) + Nx*idy + idx; 
                   
   unsigned int ix = id, 
                iy = id, 
                iz = id;


   //Dealing with boundaries
   if(idx==0){id_im = id; ix = Nx*Ny*idz + Nx*idy;} 
   if(idy==0){id_jm = id; iy = Nx*Ny*idz + idx;} 
   if(idz==0){id_km = id; iz = Nx*idy + idx;} 

   const unsigned int Offset = Nx*Ny*Nz;

   d_Phi[           id] = deltaX*(phiS[ix] - phiS[id_im]);

   d_Phi[1*Offset + id] = deltaY*(phiS[iy] - phiS[id_jm]);

   d_Phi[2*Offset + id] = deltaZ*(phiS[iz] - phiS[id_km]);
	
   return;

}

__global__ void PhiDevPlusParameter(
                                    double* const phi_xyz,
                                    const double* const d_Phi,
                                    const double* const jbn,
                                    unsigned const int Nx,
                                    unsigned const int Ny,
                                    unsigned const int Nz
                                    )
{
   unsigned const int Offset = Nx*Ny*Nz; 

   unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x,
                idy = blockDim.y*blockIdx.y + threadIdx.y, 
                idz = blockDim.z*blockIdx.z + threadIdx.z;

   unsigned int id = idx + idy*Nx + idz*Nx*Ny,
                id_im1 = (idx - 1) + idy*Nx + idz*Nx*Ny,
                id_ip1 = (idx + 1) + idy*Nx + idz*Nx*Ny,

                id_jm1 = idx + (idy - 1)*Nx + idz*Nx*Ny,
                id_jp1 = idx + (idy + 1)*Nx + idz*Nx*Ny,

                id_km1 = idx + idy*Nx + (idz - 1)*Nx*Ny,
                id_kp1 = idx + idy*Nx + (idz + 1)*Nx*Ny,

                id_im2 = (idx - 2) + idy*Nx + idz*Nx*Ny,
                id_ip2 = (idx + 2) + idy*Nx + idz*Nx*Ny,

                id_jm2 = idx + (idy - 2)*Nx + idz*Nx*Ny,
                id_jp2 = idx + (idy + 2)*Nx + idz*Nx*Ny,

                id_km2 = idx + idy*Nx + (idz - 2)*Nx*Ny,
                id_kp2 = idx + idy*Nx + (idz + 2)*Nx*Ny;

   //Dealing with boundaries

   if(idx == 0){id_im1 = id; id_im2 = id;} 
   if(idx == 1){id_im2 = id;} 
   if(idx == Nx -1){id_ip1 = id; id_ip2 = id;} 
   if(idx == Nx -2){id_ip2 = id;} 

   if(idy == 0){id_jm1 = id; id_jm2 = id;} 
   if(idy == 1){id_jm2 = id;} 
   if(idy == Ny -1){id_jp1 = id; id_jp2 = id;} 
   if(idy == Ny -2){id_jp2 = id;} 

   if(idz == 0){id_km1 = id; id_km2 = id;} 
   if(idz == 1){id_km2 = id;} 
   if(idz == Nz -1){id_kp1 = id; id_kp2 = id;} 
   if(idz == Nz -2){id_kp2 = id;} 

   double beta1, beta2, beta3, beta4, beta5;

  
   //Axis X

   beta1 = jbn[id]*d_Phi[id_im2];
   beta2 = jbn[id]*d_Phi[id_im1]; 
   beta3 = jbn[id]*d_Phi[id];
   beta4 = jbn[id]*d_Phi[id_ip1];
   beta5 = jbn[id]*d_Phi[id_ip2];

   phi_xyz[id] = Phi_x_WENO(beta1, beta2, beta3, beta4, beta5);
   //Axis Y

   beta1 = jbn[id + 4*Offset]*d_Phi[id_jm2 + 1*Offset];
   beta2 = jbn[id + 4*Offset]*d_Phi[id_jm1 + 1*Offset]; 
   beta3 = jbn[id + 4*Offset]*d_Phi[id + 1*Offset];
   beta4 = jbn[id + 4*Offset]*d_Phi[id_jp1 + 1*Offset];
   beta5 = jbn[id + 4*Offset]*d_Phi[id_jp2 + 1*Offset];


   phi_xyz[id + 1*Offset] = Phi_x_WENO(beta1, beta2, beta3, beta4, beta5);

 
   //Axis Z

   beta1 = jbn[id + 8*Offset]*d_Phi[id_km2 + 2*Offset];
   beta2 = jbn[id + 8*Offset]*d_Phi[id_km1 + 2*Offset]; 
   beta3 = jbn[id + 8*Offset]*d_Phi[id + 2*Offset];
   beta4 = jbn[id + 8*Offset]*d_Phi[id_kp1 + 2*Offset];
   beta5 = jbn[id + 8*Offset]*d_Phi[id_kp2 + 2*Offset];

   phi_xyz[id + 2*Offset] = Phi_x_WENO(beta1, beta2, beta3, beta4, beta5);
   return;
}


__global__ void PhiDevMinusParameter(
                                     double* const phi_xyz,
                                     const double* const d_Phi,
                                     const double* const jbn,
                                     unsigned const int Nx,
                                     unsigned const int Ny,
                                     unsigned const int Nz
                                    )
{
   unsigned const int Offset = Nx*Ny*Nz; 

   unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x,
                idy = blockDim.y*blockIdx.y + threadIdx.y, 
                idz = blockDim.z*blockIdx.z + threadIdx.z;

   unsigned int id = idx + idy*Nx + idz*Nx*Ny,
                id_im1 = (idx - 1) + idy*Nx + idz*Nx*Ny,
                id_ip1 = (idx + 1) + idy*Nx + idz*Nx*Ny,

                id_jm1 = idx + (idy - 1)*Nx + idz*Nx*Ny,
                id_jp1 = idx + (idy + 1)*Nx + idz*Nx*Ny,

                id_km1 = idx + idy*Nx + (idz - 1)*Nx*Ny,
                id_kp1 = idx + idy*Nx + (idz + 1)*Nx*Ny,

                id_ip2 = (idx + 2) + idy*Nx + idz*Nx*Ny,

                id_jp2 = idx + (idy + 2)*Nx + idz*Nx*Ny,

                id_kp2 = idx + idy*Nx + (idz + 2)*Nx*Ny,

                id_ip3 = (idx + 3) + idy*Nx + idz*Nx*Ny,

                id_jp3 = idx + (idy + 3)*Nx + idz*Nx*Ny,

                id_kp3 = idx + idy*Nx + (idz + 3)*Nx*Ny;

   //Dealing with boundaries

   if(idx == 0){id_im1 = id;} 
   if(idx == Nx -1){id_ip1 = id; id_ip2 = id; id_ip3 = id; } 
   if(idx == Nx -2){id_ip2 = id; id_ip3 = id; } 
   if(idx == Nx -3){id_ip3 = id; } 


   if(idy == 0){id_jm1 = id;} 
   if(idy == Ny -1){id_jp1 = id; id_jp2 = id; id_jp3 = id; } 
   if(idy == Ny -2){id_jp2 = id; id_jp3 = id; } 
   if(idy == Ny -3){id_jp3 = id; } 

   if(idz == 0){id_im1 = id;} 
   if(idz == Nz -1){id_kp1 = id; id_kp2 = id; id_kp3 = id; } 
   if(idz == Nz -2){id_kp2 = id; id_kp3 = id; } 
   if(idz == Nz -3){id_kp3 = id; } 


   double beta1, beta2, beta3, beta4, beta5;
  
   //Axis X

   beta1 = jbn[id]*d_Phi[id_ip3];
   beta2 = jbn[id]*d_Phi[id_ip2]; 
   beta3 = jbn[id]*d_Phi[id_ip1];
   beta4 = jbn[id]*d_Phi[id];
   beta5 = jbn[id]*d_Phi[id_im1];


   phi_xyz[id           ] = Phi_x_WENO(beta1, beta2, beta3, beta4, beta5);
   //Axis Y

   beta1 = jbn[id + 4*Offset]*d_Phi[id_jp3 + 1*Offset];
   beta2 = jbn[id + 4*Offset]*d_Phi[id_jp2 + 1*Offset]; 
   beta3 = jbn[id + 4*Offset]*d_Phi[id_jp1 + 1*Offset];
   beta4 = jbn[id + 4*Offset]*d_Phi[id + 1*Offset];
   beta5 = jbn[id + 4*Offset]*d_Phi[id_jm1 + 1*Offset];

   phi_xyz[id + 1*Offset] = Phi_x_WENO(beta1, beta2, beta3, beta4, beta5);

   //Axis Z


   beta1 = jbn[id + 8*Offset]*d_Phi[id_kp3 + 2*Offset];
   beta2 = jbn[id + 8*Offset]*d_Phi[id_kp2 + 2*Offset]; 
   beta3 = jbn[id + 8*Offset]*d_Phi[id_kp1 + 2*Offset];
   beta4 = jbn[id + 8*Offset]*d_Phi[id + 2*Offset];
   beta5 = jbn[id + 8*Offset]*d_Phi[id_km1 + 2*Offset];

   phi_xyz[id + 2*Offset] = Phi_x_WENO(beta1, beta2, beta3, beta4, beta5);
   return;

}





__global__ void reini_RS_WENO(
                             double* const rs,
                             const double* const phiS,                    
                             const double* const phiS0,                    
                             const double* const deltaXYZ,                    
                             const double* const d_phiP,                    
                             const double* const d_phiM,
                             unsigned int Nx,
                             unsigned int Ny,
                             unsigned int Nz 
                             )
{

   unsigned int Offset = Nx*Ny*Nz;

   unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x,
                idy = blockDim.y*blockIdx.y + threadIdx.y,
                idz = blockDim.z*blockIdx.z + threadIdx.z;

   unsigned int id = idx + Nx*idy + Nx*Ny*idz;
   double so, rs_x, rs_y, rs_z, ta, grad_mod;
   double phiMax, phiMin;

   ta = (double)(phiS[id] > 0.0) - (double)(phiS[id] < 0.0);


   //Getting gradient axis X
   phiMax = MAX(d_phiP[id   ], 0.0)*MAX(d_phiP[id   ], 0.0);  
   phiMin = MIN(d_phiM[id   ], 0.0)*MIN(d_phiM[id   ], 0.0);  

   rs_x  = 0.5*(ta + 1.0)*MAX(phiMax, phiMin);

   phiMax = MAX(d_phiM[id   ], 0.0)*MAX(d_phiM[id   ], 0.0);  
   phiMin = MIN(d_phiP[id   ], 0.0)*MIN(d_phiP[id   ], 0.0);  

   rs_x  += 0.5*abs(ta - 1.0)*MAX(phiMax, phiMin);

   //Getting gradient axis Y
   phiMax = MAX(d_phiP[id + 1*Offset], 0.0)*MAX(d_phiP[id + 1*Offset], 0.0);  
   phiMin = MIN(d_phiM[id + 1*Offset], 0.0)*MIN(d_phiM[id + 1*Offset], 0.0);  

   rs_y  = 0.5*(ta + 1.0)*MAX(phiMax, phiMin);

   phiMax = MAX(d_phiM[id + 1*Offset], 0.0)*MAX(d_phiM[id + 1*Offset], 0.0);  
   phiMin = MIN(d_phiP[id + 1*Offset], 0.0)*MIN(d_phiP[id + 1*Offset], 0.0);  

   rs_y  += 0.5*abs(ta - 1.0)*MAX(phiMax, phiMin);

   //Getting gradient axis Z
   phiMax = MAX(d_phiP[id + 2*Offset], 0.0)*MAX(d_phiP[id + 2*Offset], 0.0);  
   phiMin = MIN(d_phiM[id + 2*Offset], 0.0)*MIN(d_phiM[id + 2*Offset], 0.0);  

   rs_z  = 0.5*(ta + 1.0)*MAX(phiMax, phiMin);

   phiMax = MAX(d_phiM[id + 2*Offset], 0.0)*MAX(d_phiM[id + 2*Offset], 0.0);  
   phiMin = MIN(d_phiP[id + 2*Offset], 0.0)*MIN(d_phiP[id + 2*Offset], 0.0);  

   rs_z  += 0.5*abs(ta - 1.0)*MAX(phiMax, phiMin);

   grad_mod = sqrt(rs_x + rs_y + rs_z);

   so = phiS[id] 
      / sqrt(phiS[id]*phiS[id] + grad_mod*deltaXYZ[id]*deltaXYZ[id] );

   rs[id] = 1.0*so*(grad_mod - 1.0);

   return;
}


__global__ void advect_RS_WENO(
                              double* const rs,
                              const double* const velocity,
                              const double* const d_phiP_d,
                              const double* const d_phiM_d,
                              unsigned int Nx,
                              unsigned int Ny,
                              unsigned int Nz
                              )
{
   unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x,
                idy = blockDim.y*blockIdx.y + threadIdx.y,
                idz = blockDim.z*blockIdx.z + threadIdx.z;

   unsigned int id = idx + Nx*idy + Nx*Ny*idz,
                Offset = Nx*Ny*Nz;

   double rs_x, rs_y, rs_z;
   double grad_x, grad_y, grad_z;
   double rsign;

   rsign = (double)(velocity[id] > 0.0) 
         - (double)(velocity[id] < 0.0); 
            
   rs_x = 0.5*(rsign + 1.0)*velocity[id]*d_phiP_d[id] 
        + 0.5*abs(rsign - 1.0)*velocity[id]*d_phiM_d[id];

   grad_x = 0.5*(rsign + 1.0)*d_phiP_d[id] 
          + 0.5*abs(rsign - 1.0)*d_phiM_d[id];

   rsign = (double)(velocity[id + 1*Offset] > 0.0) 
         - (double)(velocity[id + 1*Offset] < 0.0); 

   rs_y = 0.5*(rsign + 1.0)*velocity[id + 1*Offset]*d_phiP_d[id + 1*Offset] 
        + 0.5*abs(rsign - 1.0)*velocity[id + 1*Offset]*d_phiM_d[id + 1*Offset];

   grad_y = 0.5*(rsign + 1.0)*d_phiP_d[id + 1*Offset] 
          + 0.5*abs(rsign - 1.0)*d_phiM_d[id + 1*Offset];

   rsign = (double)(velocity[id + 2*Offset] > 0.0) 
         - (double)(velocity[id + 2*Offset] < 0.0); 

   rs_z = 0.5*(rsign + 1.0)*velocity[id + 2*Offset]*d_phiP_d[id + 2*Offset] 
        + 0.5*abs(rsign - 1.0)*velocity[id + 2*Offset]*d_phiM_d[id + 2*Offset];

   grad_z = 0.5*(rsign + 1.0)*d_phiP_d[id + 2*Offset] 
          + 0.5*abs(rsign - 1.0)*d_phiM_d[id + 2*Offset];

   rs[id] = rs_x + rs_y + rs_z;
   
   return;
}






