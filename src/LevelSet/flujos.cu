#include<hip/hip_runtime.h>

__device__ double devDiv(
                         const double c1,
                         const double e1,
                         const double c2,
                         const double e2,
                         const double c3,
                         const double e3,
                         const double delta
                         )
{ 
   return delta*(c1*e1+c2*e2+c3*e3);
}
__global__ void  flux_continuity_CUDA(
                                     double* const e_flux,
                                     double* const f_flux,
                                     double* const g_flux,
                                     double* const um,
                                     unsigned const int Nx,
                                     unsigned const int Ny,
                                     unsigned const int Nz,
                                     double const c1 
                                     )
{

   unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x,
                idy = blockDim.y*blockIdx.y + threadIdx.y,
                idz = blockDim.z*blockIdx.z + threadIdx.z;

   unsigned int id = idx + idy*Nx + idz*Nx*Ny,
                Offset = Nx*Ny*Nz;

   e_flux[id] = -c1*um[id + 1*Offset];
   f_flux[id] = -c1*um[id + 2*Offset];
   g_flux[id] = -c1*um[id + 3*Offset];

   //Viscous terms
   e_flux[id + Offset] = 0.0;
   f_flux[id + Offset] = 0.0;
   g_flux[id + Offset] = 0.0;

   return;
}

__global__ void  flux_momentumX_CUDA(
                                     double* const e_flux,
                                     double* const f_flux,
                                     double* const g_flux,
                                     double* const u,
                                     double* const um,
                                     double* const press,
                                     double* const dcvel,
                                     double* const ddvelp,
                                     double* const vis,
                                     double* const jbn,
                                     unsigned const int Nx,
                                     unsigned const int Ny,
                                     unsigned const int Nz,
                                     double const cdiv 
                                     )
{

   unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x,
                idy = blockDim.y*blockIdx.y + threadIdx.y,
                idz = blockDim.z*blockIdx.z + threadIdx.z;

   unsigned int id = idx + idy*Nx + idz*Nx*Ny,
                Offset = Nx*Ny*Nz;

   double param0, param1, param2;

   e_flux[id] = -um[id + 1*Offset]*u[id] - press[id];
   f_flux[id] = -um[id + 2*Offset]*u[id];
   g_flux[id] = -um[id + 3*Offset]*u[id];


   //Viscous terms
   param0 = jbn[id           ]*ddvelp[id           ]   
          + jbn[id + 3*Offset]*ddvelp[id + 3*Offset]
          + jbn[id + 6*Offset]*ddvelp[id + 6*Offset];

   param1 = jbn[id + 1*Offset]*dcvel[id + 1*Offset]   
          + jbn[id + 4*Offset]*dcvel[id + 4*Offset]
          + jbn[id + 7*Offset]*dcvel[id + 7*Offset];

   param2 = jbn[id + 2*Offset]*dcvel[id + 2*Offset]   
          + jbn[id + 5*Offset]*dcvel[id + 5*Offset]
          + jbn[id + 8*Offset]*dcvel[id + 8*Offset];

   e_flux[id + Offset] = vis[id]*cdiv*(2.0*param0 - param1 - param2);

   param1 = jbn[id           ]*ddvelp[id + 1*Offset]   
          + jbn[id + 1*Offset]*ddvelp[id + 4*Offset]
          + jbn[id + 2*Offset]*ddvelp[id + 7*Offset];

   param2 = jbn[id + 1*Offset]*dcvel[id           ]   
          + jbn[id + 4*Offset]*dcvel[id + 3*Offset]
          + jbn[id + 7*Offset]*dcvel[id + 6*Offset];

   f_flux[id + Offset] = vis[id]*(param1 + param2);

   param1 = jbn[id           ]*dcvel[id + 2*Offset]   
          + jbn[id + 1*Offset]*dcvel[id + 5*Offset]
          + jbn[id + 2*Offset]*dcvel[id + 8*Offset];

   param2 = jbn[id + 2*Offset]*ddvelp[id           ]   
          + jbn[id + 5*Offset]*ddvelp[id + 3*Offset]
          + jbn[id + 8*Offset]*ddvelp[id + 6*Offset];

   g_flux[id + Offset] = vis[id]*(param1 + param2);

   return;
}


__global__ void  flux_momentumY_CUDA(
                                     double* const e_flux,
                                     double* const f_flux,
                                     double* const g_flux,
                                     double* const u,
                                     double* const um,
                                     double* const press,
                                     double* const dcvel,
                                     double* const ddvelp,
                                     double* const vis,
                                     double* const jbn,
                                     unsigned const int Nx,
                                     unsigned const int Ny,
                                     unsigned const int Nz,
                                     double const cdiv 
                                     )
{

   unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x,
                idy = blockDim.y*blockIdx.y + threadIdx.y,
                idz = blockDim.z*blockIdx.z + threadIdx.z;

   unsigned int id = idx + idy*Nx + idz*Nx*Ny,
                Offset = Nx*Ny*Nz;

   double param0, param1, param2;

   e_flux[id] = -um[id + 1*Offset]*u[id + 1*Offset];
   f_flux[id] = -um[id + 2*Offset]*u[id + 1*Offset] -  press[id];
   g_flux[id] = -um[id + 3*Offset]*u[id + 1*Offset];


   //Viscous terms

   param1 = jbn[id           ]*ddvelp[id + 1*Offset]   
          + jbn[id + 3*Offset]*ddvelp[id + 4*Offset]
          + jbn[id + 6*Offset]*ddvelp[id + 7*Offset];

   param2 = jbn[id + 1*Offset]*dcvel[id + 0*Offset]   
          + jbn[id + 4*Offset]*dcvel[id + 3*Offset]
          + jbn[id + 7*Offset]*dcvel[id + 6*Offset];

   e_flux[id + Offset] = vis[id]*(param1 + param2);


   param0 = jbn[id + 0*Offset]*dcvel[id + 0*Offset]   
          + jbn[id + 3*Offset]*dcvel[id + 3*Offset]
          + jbn[id + 6*Offset]*dcvel[id + 6*Offset];

   param1 = jbn[id + 1*Offset]*ddvelp[id + 1*Offset]   
          + jbn[id + 4*Offset]*ddvelp[id + 4*Offset]
          + jbn[id + 7*Offset]*ddvelp[id + 7*Offset];

   param2 = jbn[id + 2*Offset]*dcvel[id + 2*Offset]   
          + jbn[id + 5*Offset]*dcvel[id + 5*Offset]
          + jbn[id + 8*Offset]*dcvel[id + 8*Offset];

   f_flux[id + Offset] = vis[id]*cdiv*(2.0*param1 - param0 - param2);

   param1 = jbn[id + 1*Offset]*dcvel[id + 2*Offset]   
          + jbn[id + 4*Offset]*dcvel[id + 5*Offset]
          + jbn[id + 7*Offset]*dcvel[id + 8*Offset];

   param2 = jbn[id + 2*Offset]*ddvelp[id + 1*Offset]   
          + jbn[id + 5*Offset]*ddvelp[id + 4*Offset]
          + jbn[id + 8*Offset]*ddvelp[id + 7*Offset];

   g_flux[id + Offset] = vis[id]*(param1 + param2);

   return;
}


__global__ void  flux_momentumZ_CUDA(
                                     double* const e_flux,
                                     double* const f_flux,
                                     double* const g_flux,
                                     double* const u,
                                     double* const um,
                                     double* const press,
                                     double* const dcvel,
                                     double* const ddvelp,
                                     double* const vis,
                                     double* const jbn,
                                     double* const xmesh,
                                     unsigned const int Nx,
                                     unsigned const int Ny,
                                     unsigned const int Nz,
                                     double const cdiv,
                                     double const froude
                                     )
{

   unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x,
                idy = blockDim.y*blockIdx.y + threadIdx.y,
                idz = blockDim.z*blockIdx.z + threadIdx.z;

   unsigned int id = idx + idy*Nx + idz*Nx*Ny,
                Offset = Nx*Ny*Nz;

   double param0, param1, param2;

   e_flux[id] = -um[id + 1*Offset]*u[id + 2*Offset];
   f_flux[id] = -um[id + 2*Offset]*u[id + 2*Offset];
   g_flux[id] = -um[id + 3*Offset]*u[id + 2*Offset] - press[id] 
                -froude*xmesh[id + 2*Offset];


   //Viscous terms
   param1 = jbn[id           ]*ddvelp[id + 2*Offset]   
          + jbn[id + 3*Offset]*ddvelp[id + 5*Offset]
          + jbn[id + 6*Offset]*ddvelp[id + 8*Offset];

   param2 = jbn[id + 2*Offset]*dcvel[id + 0*Offset]   
          + jbn[id + 5*Offset]*dcvel[id + 3*Offset]
          + jbn[id + 8*Offset]*dcvel[id + 6*Offset];

   e_flux[id + Offset] = vis[id]*(param1 + param2);


   param1 = jbn[id + 2*Offset]*dcvel[id + 1*Offset]   
          + jbn[id + 5*Offset]*dcvel[id + 4*Offset]
          + jbn[id + 8*Offset]*dcvel[id + 7*Offset];

   param2 = jbn[id + 1*Offset]*ddvelp[id + 2*Offset]   
          + jbn[id + 4*Offset]*ddvelp[id + 5*Offset]
          + jbn[id + 7*Offset]*ddvelp[id + 8*Offset];


   f_flux[id + Offset] = vis[id]*(param1 + param2);

   param0 = jbn[id + 0*Offset]*dcvel[id + 0*Offset]   
          + jbn[id + 3*Offset]*dcvel[id + 3*Offset]
          + jbn[id + 6*Offset]*dcvel[id + 6*Offset];

   param1 = jbn[id + 1*Offset]*dcvel[id + 1*Offset]   
          + jbn[id + 4*Offset]*dcvel[id + 4*Offset]
          + jbn[id + 7*Offset]*dcvel[id + 7*Offset];

   param2 = jbn[id + 2*Offset]*ddvelp[id + 2*Offset]   
          + jbn[id + 5*Offset]*ddvelp[id + 5*Offset]
          + jbn[id + 8*Offset]*ddvelp[id + 8*Offset];

   g_flux[id + Offset] = vis[id]*cdiv*(2.0*param2 - param1 - param0);

   return;

}


__global__ void  flux_Energy_CUDA(
                                  double* const e_flux,
                                  double* const f_flux,
                                  double* const g_flux,
                                  double* const u,
                                  double* const um,
                                  double* const dTemp,
                                  double* const vis5,
                                  double* const jbn,
                                  unsigned const int Nx,
                                  unsigned const int Ny,
                                  unsigned const int Nz
                                  )
{

   unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x,
                idy = blockDim.y*blockIdx.y + threadIdx.y,
                idz = blockDim.z*blockIdx.z + threadIdx.z;

   unsigned int id = idx + idy*Nx + idz*Nx*Ny,
                Offset = Nx*Ny*Nz;

   e_flux[id] = -um[id + 4*Offset]*u[id + 0*Offset];
   f_flux[id] = -um[id + 4*Offset]*u[id + 1*Offset];
   g_flux[id] = -um[id + 4*Offset]*u[id + 2*Offset];


   //Viscous terms
   e_flux[id + Offset] = vis5[id]*(
                         jbn[id           ]*dTemp[id + 0*Offset]   
                       + jbn[id + 3*Offset]*dTemp[id + 1*Offset]
                       + jbn[id + 6*Offset]*dTemp[id + 2*Offset]);

   f_flux[id + Offset] = vis5[id]*(
                         jbn[id + 1*Offset]*dTemp[id + 0*Offset]   
                       + jbn[id + 4*Offset]*dTemp[id + 1*Offset]
                       + jbn[id + 7*Offset]*dTemp[id + 2*Offset]);

   g_flux[id + Offset] = vis5[id]*(
                         jbn[id + 2*Offset]*dTemp[id + 0*Offset]   
                       + jbn[id + 5*Offset]*dTemp[id + 1*Offset]
                       + jbn[id + 8*Offset]*dTemp[id + 2*Offset]);


   return;

}

