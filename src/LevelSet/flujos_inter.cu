#include<hip/hip_runtime.h>
#include<stdio.h>
#include"flujos.h"
#include"divergencia.h"
#include"extrapol.h"

void rsDivergence(
                 double* const rs_d,
                 double* const e_d,
                 double* const f_d,
                 double* const g_d,
                 double* const ex_d,
                 double* const fy_d,
                 double* const gz_d,
                 const double* const jbn_d,
                 unsigned int Nx,
                 unsigned int Ny,
                 unsigned int Nz,
                 const double deltaX,
                 const double deltaY,
                 const double deltaZ,
                 unsigned int ncp,
                 unsigned int itera
                 );


extern "C"{
void flujos(
            double* const um1,
            double* const um,
            double* const um02,
            const double* const u,
            const double* const ug,
            const double* const dcvel,
            const double* const dcvelg,
            const double* const ddvelp,
            const double* const ddvelpg,
            const double* const press,
            const double* const pressg,
            const double* const dTemp,
            const double* const dTempg,
            const double* const vis,
            const double* const visg,
            const double* const vis5,
            const double* const vis5g,
            const double* const jbn_p,
            const double* const jbn_n,
            const double* const xmesh,
            const double c1, 
            const double dt, 
            const double froude, 
            const double deltaX, 
            const double deltaY, 
            const double deltaZ, 
            const unsigned int Nx, const unsigned int Ny, const unsigned int Nz,
            int itera, int ncp
            )
{


   double *um_d, *um1_d, *um02_d, *u_d, *ug_d, *dcvel_d, *dcvelg_d,
          *ddvelp_d, *ddvelpg_d, *press_d, *pressg_d, *dTemp_d, *dTempg_d, 
          *vis_d, *visg_d, *vis5_d, *vis5g_d, *jbn_d, *xmesh_d;
  
   double *e_d, *f_d, *g_d; 
   double *ex_d, *fy_d, *gz_d, *rs_d; 

   unsigned int Offset = Nx*Ny*Nz;

   hipMalloc((void**)&um_d, sizeof(double)*Offset*10);
   hipMalloc((void**)&um1_d, sizeof(double)*Offset*10);
   hipMalloc((void**)&um02_d, sizeof(double)*Offset*10);
   hipMalloc((void**)&u_d, sizeof(double)*Offset*3);
   hipMalloc((void**)&ug_d, sizeof(double)*Offset*3);
   hipMalloc((void**)&dcvel_d, sizeof(double)*Offset*9);
   hipMalloc((void**)&dcvelg_d, sizeof(double)*Offset*9);
   hipMalloc((void**)&ddvelp_d, sizeof(double)*Offset*9);
   hipMalloc((void**)&ddvelpg_d, sizeof(double)*Offset*9);
   hipMalloc((void**)&dTemp_d, sizeof(double)*Offset*3);
   hipMalloc((void**)&dTempg_d, sizeof(double)*Offset*3);
   hipMalloc((void**)&press_d, sizeof(double)*Offset);
   hipMalloc((void**)&pressg_d, sizeof(double)*Offset);
   hipMalloc((void**)&vis_d, sizeof(double)*Offset);
   hipMalloc((void**)&visg_d, sizeof(double)*Offset);
   hipMalloc((void**)&vis5_d, sizeof(double)*Offset);
   hipMalloc((void**)&vis5g_d, sizeof(double)*Offset);
   hipMalloc((void**)&xmesh_d, sizeof(double)*3*Offset);

   hipMalloc((void**)&e_d, sizeof(double)*2*Offset);
   hipMalloc((void**)&f_d, sizeof(double)*2*Offset);
   hipMalloc((void**)&g_d, sizeof(double)*2*Offset);

   hipMalloc((void**)&ex_d, sizeof(double)*2*Offset);
   hipMalloc((void**)&fy_d, sizeof(double)*2*Offset);
   hipMalloc((void**)&gz_d, sizeof(double)*2*Offset);
   hipMalloc((void**)&rs_d, sizeof(double)*Offset);

   hipMalloc((void**)&jbn_d, sizeof(double)*Offset*11);

   hipMemcpy(um_d, um,  sizeof(double)*Offset*10, hipMemcpyHostToDevice);
   hipMemcpy(um02_d, um02,  sizeof(double)*Offset*10, hipMemcpyHostToDevice);
   hipMemcpy(u_d, u,  sizeof(double)*Offset*3, hipMemcpyHostToDevice);
   hipMemcpy(ug_d, ug,  sizeof(double)*Offset*3, hipMemcpyHostToDevice);
   hipMemcpy(dcvel_d, dcvel,  sizeof(double)*Offset*9, hipMemcpyHostToDevice);
   hipMemcpy(dcvelg_d, dcvelg,  sizeof(double)*Offset*9, hipMemcpyHostToDevice);
   hipMemcpy(ddvelp_d, ddvelp,  sizeof(double)*Offset*9, hipMemcpyHostToDevice);
   hipMemcpy(ddvelpg_d, ddvelpg,  sizeof(double)*Offset*9, hipMemcpyHostToDevice);
   hipMemcpy(press_d, press,  sizeof(double)*Offset, hipMemcpyHostToDevice);
   hipMemcpy(pressg_d, pressg,  sizeof(double)*Offset, hipMemcpyHostToDevice);
   hipMemcpy(dTemp_d, dTemp,  sizeof(double)*Offset*3, hipMemcpyHostToDevice);
   hipMemcpy(dTempg_d, dTempg,  sizeof(double)*Offset*3, hipMemcpyHostToDevice);
   hipMemcpy(vis_d, vis,  sizeof(double)*Offset, hipMemcpyHostToDevice);
   hipMemcpy(visg_d, visg,  sizeof(double)*Offset, hipMemcpyHostToDevice);
   hipMemcpy(vis5_d, vis5,  sizeof(double)*Offset, hipMemcpyHostToDevice);
   hipMemcpy(vis5g_d, vis5g,  sizeof(double)*Offset, hipMemcpyHostToDevice);
   hipMemcpy(xmesh_d, xmesh,  sizeof(double)*3*Offset, hipMemcpyHostToDevice);


   const double cdiv = 2.0/3.0;
   
   if((ncp + itera)%2){
     hipMemcpy(jbn_d, jbn_n,  sizeof(double)*Offset*11, hipMemcpyHostToDevice);
   }
   else{
     hipMemcpy(jbn_d, jbn_p,  sizeof(double)*Offset*11, hipMemcpyHostToDevice);
   }

   dim3 DimGrid(12,12,24);
   dim3 DimBlock(8,8,4);


   for(int num_flujo = 0; num_flujo <= 9 ; num_flujo++){

      if(num_flujo == 0){//continuity liquid
        flux_continuity_CUDA<<<DimGrid, DimBlock>>>( e_d, f_d , g_d, um_d, Nx, Ny, Nz, c1);
      }

      
      else if(num_flujo == 1){//momentum X liquid
         flux_momentumX_CUDA<<<DimGrid, DimBlock>>>(e_d, f_d, g_d, 
                                                    u_d, um_d, press_d, dcvel_d, ddvelp_d,
                                                    vis_d, jbn_d, Nx, Ny, Nz, cdiv);
      }

      else if(num_flujo == 2){//momentum Y liquid
         flux_momentumY_CUDA<<<DimGrid, DimBlock>>>(e_d, f_d, g_d, 
                                                    u_d, um_d, press_d, dcvel_d, ddvelp_d,
                                                    vis_d, jbn_d, Nx, Ny, Nz, cdiv);
      }


      else if(num_flujo == 3){//momentum Z liquid
         flux_momentumZ_CUDA<<<DimGrid, DimBlock>>>(e_d, f_d, g_d,
                                                    u_d, um_d, press_d, dcvel_d,
                                                    ddvelp_d, vis_d, jbn_d, xmesh_d, 
                                                    Nx, Ny, Nz, cdiv, froude);
      }


      else if(num_flujo == 4){//Energy liquid
           flux_Energy_CUDA<<<DimGrid, DimBlock>>>(e_d, f_d, g_d,
                                                   u_d, um_d, dTemp_d, vis5_d, jbn_d,
                                                   Nx, Ny, Nz);
      }

      else if(num_flujo == 5){//continuity gas
        flux_continuity_CUDA<<<DimGrid, DimBlock>>>( e_d, f_d , g_d, &(um_d[5*Offset]), Nx, Ny, Nz, c1);
      }

      else if(num_flujo == 6){//momentum X gas
         flux_momentumX_CUDA<<<DimGrid, DimBlock>>>(e_d, f_d, g_d, 
                                             ug_d, &(um_d[5*Offset]), pressg_d, dcvelg_d, ddvelpg_d,
                                              visg_d, jbn_d, Nx, Ny, Nz, cdiv);
      }

      else if(num_flujo == 7){//momentum Y gas
         flux_momentumX_CUDA<<<DimGrid, DimBlock>>>(e_d, f_d, g_d, 
                             ug_d, &(um_d[5*Offset]), pressg_d, dcvelg_d, ddvelpg_d,
                             visg_d, jbn_d, Nx, Ny, Nz, cdiv);
      }

      else if(num_flujo == 8){//momentum Z liquid
         flux_momentumZ_CUDA<<<DimGrid, DimBlock>>>(e_d, f_d, g_d,
                                                    ug_d, &(um_d[5*Offset]), pressg_d, dcvelg_d,
                                                    ddvelpg_d, visg_d, jbn_d, xmesh_d, 
                                                    Nx, Ny, Nz, cdiv, 0.0);
      }

      else if(num_flujo == 9){//Energy liquid
           flux_Energy_CUDA<<<DimGrid, DimBlock>>>(e_d, f_d, g_d,
                                                   ug_d, &(um_d[5*Offset]), dTempg_d, vis5g_d, jbn_d,
                                                   Nx, Ny, Nz);
      }
    
      rsDivergence(rs_d, e_d, f_d, g_d, ex_d, fy_d, gz_d, jbn_d, Nx, Ny, Nz, 
                   deltaX, deltaY, deltaZ, ncp, itera);
   
      if(ncp==1){
          RunGK_FirstS<<<DimGrid,DimBlock>>>(&(um1_d[num_flujo*Offset]), &(um_d[num_flujo*Offset]), 
                                             dt, rs_d, Nx, Ny, Nz);
      }
      else if(ncp==2){
          RunGK_SecondS<<<DimGrid,DimBlock>>>(&(um1_d[num_flujo*Offset]), &(um02_d[num_flujo*Offset]), 
                                              &(um_d[num_flujo*Offset]), dt, rs_d, Nx, Ny, Nz);
      }

 
   }

   hipMemcpy(um1, um1_d,  sizeof(double)*Offset*10, hipMemcpyDeviceToHost);

   // check for error
   hipError_t error = hipGetLastError();
   if(error != hipSuccess)
   {   
   // print the CUDA error message and exit
      printf("CUDA error: %s\n", hipGetErrorString(error));
      exit(-1);
   }   
    
 
   return;
}
}

void rsDivergence(
                 double* const rs_d,
                 double* const e_d,
                 double* const f_d,
                 double* const g_d,
                 double* const ex_d,
                 double* const fy_d,
                 double* const gz_d,
                 const double* const jbn_d,
                 unsigned int Nx,
                 unsigned int Ny,
                 unsigned int Nz,
                 const double deltaX,
                 const double deltaY,
                 const double deltaZ,
                 unsigned int ncp,
                 unsigned int itera
                 )
{

   unsigned int Offset  = Nx*Ny*Nz;
 
   dim3 DimGrid(12,12,24);
   dim3 DimBlock(8,8,4);

   //Convective fluxes 
   get_flux_e_CUDA<<<DimGrid, DimBlock>>>(ex_d, e_d, f_d, g_d, jbn_d, Nx, Ny, Nz);       
   get_flux_f_CUDA<<<DimGrid, DimBlock>>>(fy_d, e_d, f_d, g_d, jbn_d, Nx, Ny, Nz);       
   get_flux_g_CUDA<<<DimGrid, DimBlock>>>(gz_d, e_d, f_d, g_d, jbn_d, Nx, Ny, Nz);       

   //Diffusive fluxes
   get_flux_e_CUDA<<<DimGrid, DimBlock>>>(&(ex_d[Offset]), &(e_d[Offset]), &(f_d[Offset]), 
                                          &(g_d[Offset]), jbn_d, Nx, Ny, Nz);       
   get_flux_f_CUDA<<<DimGrid, DimBlock>>>(&(fy_d[Offset]), &(e_d[Offset]), &(f_d[Offset]), 
                                          &(g_d[Offset]), jbn_d, Nx, Ny, Nz);       
   get_flux_g_CUDA<<<DimGrid, DimBlock>>>(&(gz_d[Offset]), &(e_d[Offset]), &(f_d[Offset]), 
                                          &(g_d[Offset]), jbn_d, Nx, Ny, Nz);       
                   

   double cons1 = 7.0 / 6.0;
   double cons2 = -8.0 / 6.0;
   double cons3 = 1.0 / 6.0;

   if((ncp + itera)%2){

      divDevXMin<<<DimGrid, DimBlock>>>( e_d, ex_d, Nx, Ny, Nz, deltaX, cons1, cons2, cons3);
      divDevXMin<<<DimGrid, DimBlock>>>( &(e_d[Offset]), &(ex_d[Offset]), Nx, Ny, Nz, deltaX, cons1, cons2, cons3);
 
      divDevYMin<<<DimGrid, DimBlock>>>( f_d, fy_d, Nx, Ny, Nz, deltaY, cons1, cons2, cons3);
      divDevYMin<<<DimGrid, DimBlock>>>( &(f_d[Offset]), &(fy_d[Offset]), Nx, Ny, Nz, deltaY, cons1, cons2, cons3);
 
      divDevZMin<<<DimGrid, DimBlock>>>( g_d, gz_d, Nx, Ny, Nz, deltaZ, cons1, cons2, cons3);
      divDevZMin<<<DimGrid, DimBlock>>>( &(g_d[Offset]), &(gz_d[Offset]), Nx, Ny, Nz, deltaZ, cons1, cons2, cons3);

   }
   else{

      divDevXPlus<<<DimGrid, DimBlock>>>( e_d, ex_d, Nx, Ny, Nz, deltaX, cons1, cons2, cons3);
      divDevXPlus<<<DimGrid, DimBlock>>>( &(e_d[Offset]), &(ex_d[Offset]), Nx, Ny, Nz, deltaX, cons1, cons2, cons3);
 
      divDevYPlus<<<DimGrid, DimBlock>>>( f_d, fy_d, Nx, Ny, Nz, deltaY, cons1, cons2, cons3);
      divDevYPlus<<<DimGrid, DimBlock>>>( &(f_d[Offset]), &(fy_d[Offset]), Nx, Ny, Nz, deltaY, cons1, cons2, cons3);
 
      divDevZPlus<<<DimGrid, DimBlock>>>( g_d, gz_d, Nx, Ny, Nz, deltaZ, cons1, cons2, cons3);
      divDevZPlus<<<DimGrid, DimBlock>>>( &(g_d[Offset]), &(gz_d[Offset]), Nx, Ny, Nz, deltaZ, cons1, cons2, cons3);

   }

   rs_divergence_CUDA<<<DimGrid, DimBlock>>>(rs_d, e_d, f_d, g_d, jbn_d, Nx, Ny, Nz); 

   // check for error
   hipError_t error = hipGetLastError();
   if(error != hipSuccess)
   {   
   // print the CUDA error message and exit
      printf("CUDA error: %s\n", hipGetErrorString(error));
      exit(-1);
   }   
   
   return;
}

