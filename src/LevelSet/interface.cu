#include<hip/hip_runtime.h>
#include<stdio.h>
#include "LevelSet/extrapol.h"
#include "LevelSet/dimdef.h"


void exta(
         double* extVal_d,
         double* phiS_d,
         double* jbn_d,
         double* d_Phi_d,
         double* rs_d,
         double* d_d,
         double deltaX, double deltaY, double deltaZ,
         int Nx, int Ny, int Nz,
         double dtext,
         int Flag);

extern "C"{


void callCUDA(
             double* pressg,
             double* velocity,
             double* phiS,
             double* jbn,
             double deltaX,
             double deltaY,
             double deltaZ,
             unsigned int Nx,
             unsigned int Ny,
             unsigned int Nz,
             double dtext
             )
{

   unsigned int Offset = Nx*Ny*Nz;
   double *pressg_d, *velocity_d, *phiS_d, *jbn_d,
          *rs_d, *d_Phi_d, *extVal_d,
          *d_d;

   hipMalloc((void**)&pressg_d,sizeof(double)*Offset);
   hipMalloc((void**)&velocity_d,sizeof(double)*3*Offset);
   hipMalloc((void**)&phiS_d,sizeof(double)*Offset);
   hipMalloc((void**)&jbn_d,sizeof(double)*11*Offset);

   hipMalloc((void**)&d_d,sizeof(double)*Offset);


   hipMalloc((void**)&extVal_d,sizeof(double)*Offset);
   hipMalloc((void**)&d_Phi_d,sizeof(double)*3*Offset);
   hipMalloc((void**)&rs_d,sizeof(double)*Offset);

   hipMemcpy(pressg_d, pressg, sizeof(double)*Offset, 
              hipMemcpyHostToDevice );

   hipMemcpy(velocity_d,velocity,sizeof(double)*3*Offset,
              hipMemcpyHostToDevice );

   hipMemcpy(phiS_d, phiS, sizeof(double)*Offset, 
              hipMemcpyHostToDevice );

   hipMemcpy(jbn_d, jbn, sizeof(double)*11*Offset, 
              hipMemcpyHostToDevice );

   dim3 DimBlock(BLOCKDMX,BLOCKDMY,BLOCKDMZ);   
   dim3 DimGrid(GRIDMX,GRIDMY,GRIDMZ);   

   DevFirstOrder_LS<<<DimGrid, DimBlock>>>(
                                          d_Phi_d,
                                          phiS_d,
                                          jbn_d,
                                          deltaX,
                                          deltaY,
                                          deltaZ,
                                          Nx,
                                          Ny,
                                          Nz 
                                          );

  


// Extrapolating Velocity liquid variables

   exta(velocity_d, phiS_d, jbn_d, d_Phi_d, rs_d, d_d,
        deltaX, deltaY, deltaZ, Nx, Ny, Nz,
        dtext, -1);

       printf("  U Velocity Liquid \n");

   exta(&(velocity_d[1*Offset]), phiS_d, jbn_d, d_Phi_d, rs_d, d_d,
        deltaX, deltaY, deltaZ, Nx, Ny, Nz,
        dtext, -1);

       printf("  V Velocity Liquid \n");

   exta(&(velocity_d[2*Offset]), phiS_d, jbn_d, d_Phi_d, rs_d, d_d,
        deltaX, deltaY, deltaZ, Nx, Ny, Nz,
        dtext, -1);

       printf("  W Velocity Liquid \n");

// Extrapolating Gas Pressure Variable

   exta(pressg_d, phiS_d, jbn_d, d_Phi_d, rs_d, d_d,
        deltaX, deltaY, deltaZ, Nx, Ny, Nz,
        dtext, 1);

       printf("  Pressure Gas \n");

// Returning values from Device to Host

   hipMemcpy(velocity,velocity_d,sizeof(double)*3*Offset,
              hipMemcpyDeviceToHost );

   hipMemcpy(pressg,pressg_d,sizeof(double)*Offset,
              hipMemcpyDeviceToHost );
    
   hipFree(pressg_d);
   hipFree(velocity_d);
   hipFree(jbn_d);
   hipFree(phiS_d);
   hipFree(d_Phi_d);
   hipFree(d_d);
   hipFree(extVal_d);
   hipFree(rs_d);

   return;
}

}

extern "C"{

void normalLEvelSetCUDA(
                       double* nV,
                       double* phiS,
                       double* jbn,
                       double deltaX,
                       double deltaY,
                       double deltaZ,
                       unsigned int Nx,
                       unsigned int Ny,
                       unsigned int Nz
                       )
{

   unsigned int Offset = Nx*Ny*Nz;

   double *nV_d, *phiS_d, *jbn_d;

   hipMalloc((void**)&nV_d,sizeof(double)*3*Offset);
   hipMalloc((void**)&phiS_d,sizeof(double)*Offset);
   hipMalloc((void**)&jbn_d,sizeof(double)*11*Offset);

   hipMemcpy(phiS_d, phiS, sizeof(double)*Offset, 
              hipMemcpyHostToDevice );

   hipMemcpy(jbn_d, jbn, sizeof(double)*11*Offset, 
              hipMemcpyHostToDevice );

   dim3 DimBlock(BLOCKDMX,BLOCKDMY,BLOCKDMZ);   
   dim3 DimGrid(GRIDMX,GRIDMY,GRIDMZ);   

   DevFirstOrder_LS<<<DimGrid, DimBlock>>>(
                                          nV_d,
                                          phiS_d,
                                          jbn_d,
                                          deltaX,
                                          deltaY,
                                          deltaZ,
                                          Nx,
                                          Ny,
                                          Nz 
                                          );

  

// Returning values from Device to Host

   hipMemcpy(nV,nV_d,sizeof(double)*3*Offset,
              hipMemcpyDeviceToHost );
    
   hipFree(jbn_d);
   hipFree(phiS_d);
   hipFree(nV_d);

   return;
}

}


extern "C"{

void extrapolVarCUDA(
                    double* valToExt,
                    double* phiS,
                    double* d_Phi,
                    double* jbn,
                    double deltaX,
                    double deltaY,
                    double deltaZ,
                    unsigned int Nx,
                    unsigned int Ny,
                    unsigned int Nz,
                    double dtext
                    )
{

   unsigned int Offset = Nx*Ny*Nz;
   double *valToExt_d, *phiS_d, *jbn_d,
          *rs_d, *d_Phi_d, *extVal_d,
          *d_d;

   hipMalloc((void**)&valToExt_d,sizeof(double)*Offset);
   hipMalloc((void**)&phiS_d,sizeof(double)*Offset);
   hipMalloc((void**)&d_Phi_d,sizeof(double)*3*Offset);
   hipMalloc((void**)&jbn_d,sizeof(double)*11*Offset);

   hipMalloc((void**)&extVal_d,sizeof(double)*Offset);
   hipMalloc((void**)&rs_d,sizeof(double)*Offset);
   hipMalloc((void**)&d_d,sizeof(double)*Offset);

   hipMemcpy(valToExt_d,valToExt,sizeof(double)*Offset,
              hipMemcpyHostToDevice );

   hipMemcpy(phiS_d, phiS, sizeof(double)*Offset, 
              hipMemcpyHostToDevice );

   hipMemcpy(jbn_d, jbn, sizeof(double)*11*Offset, 
              hipMemcpyHostToDevice );

   hipMemcpy(d_Phi_d,d_Phi,sizeof(double)*3*Offset,
              hipMemcpyHostToDevice );

   dim3 DimBlock(BLOCKDMX,BLOCKDMY,BLOCKDMZ);   
   dim3 DimGrid(GRIDMX,GRIDMY,GRIDMZ);   


// Extrapolating Velocity liquid variables

   exta(valToExt_d, phiS_d, jbn_d, d_Phi_d, rs_d, d_d,
        deltaX, deltaY, deltaZ, Nx, Ny, Nz,
        dtext, 1);

   printf("  Ext-Some Val \n");

// Returning values from Device to Host

   hipMemcpy(valToExt,valToExt_d,sizeof(double)*Offset,
              hipMemcpyDeviceToHost );
    
   hipFree(valToExt_d);
   hipFree(jbn_d);
   hipFree(phiS_d);
   hipFree(d_Phi_d);
   hipFree(d_d);
   hipFree(extVal_d);
   hipFree(rs_d);

   return;
}

}



void exta(
         double* extVal_d,
         double* phiS_d,
         double* jbn_d,
         double* d_Phi_d,
         double* rs_d,
         double* d_d,
         double deltaX, double deltaY, double deltaZ,
         int Nx, int Ny, int Nz,
         double dtext,
         int Flag
         )
{


   dim3 DimBlock(BLOCKDMX,BLOCKDMY,BLOCKDMZ);   
   dim3 DimGrid(GRIDMX,GRIDMY,GRIDMZ);   
   printf("\n\n  Extrapolating on CUDA Device: \n");

   for(int itera = 1 ; itera <=10 ; itera++){

       extrapolKernel<<<DimGrid, DimBlock>>>(
                                          rs_d,         
                                          extVal_d, phiS_d, jbn_d, d_Phi_d,
                                            deltaX, deltaY, deltaZ,
                                            Nx, Ny, Nz, 
                                            Flag
                                            );  
     
       RunGK_FirstS<<<DimGrid, DimBlock>>>( d_d, extVal_d, 
                                           dtext, rs_d, Nx, Ny, Nz);
     
     
       extrapolKernel<<<DimGrid, DimBlock>>>(
                                            rs_d, d_d, 
                                            phiS_d, jbn_d, d_Phi_d,
                                            deltaX, deltaY, deltaZ,
                                            Nx, Ny, Nz, 
                                            Flag);
     
       RunGK_SecondS<<<DimGrid, DimBlock>>>( d_d, 
                                             extVal_d, d_d, 
                                             dtext, rs_d, 
                                             Nx, Ny, Nz);
     
     
       extrapolKernel<<<DimGrid, DimBlock>>>(
                                            rs_d, d_d, 
                                            phiS_d, jbn_d, d_Phi_d,
                                            deltaX, deltaY, deltaZ,
                                            Nx, Ny, Nz, 
                                            Flag);
     
       RunGK_ThirdS<<<DimGrid, DimBlock>>>(  extVal_d, 
                                             extVal_d, d_d, 
                                             dtext, rs_d, 
                                             Nx, Ny, Nz);
       
   }
 // check for error
  hipError_t error = hipGetLastError();
  if(error != hipSuccess)
  {
    // print the CUDA error message and exit
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }
 
 }

 
void cuExtrapolation(
                    double* extVal_d,
                    double* phiS_d,
                    double* jbn_d,
                    double deltaX, double deltaY, double deltaZ,
                    int Nx, int Ny, int Nz,
                    double dtext,
                    int Flag
                    )
{

   double *d_dPhi;
   double *rs_d;
   double *d_d;

   int Offset = Nx*Ny*Nz;

   hipMalloc((void**)&d_dPhi, 3*sizeof(double)*Offset);
   hipMalloc((void**)&rs_d,     sizeof(double)*Offset);
   hipMalloc((void**)&d_d,      sizeof(double)*Offset);

   int numGBX, numGBY,numGBZ;

   dim3 dimBlock(10,10,5);   

   numGBX = Nx / 10;
   numGBY = Ny / 10;
   numGBZ = Nz / 5;

   dim3 dimGrid(numGBX,numGBY,numGBZ);   
   

   DevFirstOrder_LS<<<dimGrid, dimBlock>>>(
                                          d_dPhi,
                                          phiS_d,
                                          jbn_d,
                                          1.0/deltaX,
                                          1.0/deltaY,
                                          1.0/deltaZ,
                                          Nx,
                                          Ny,
                                          Nz 
                                          );
   printf("\n\n  Extrapolating on CUDA Device: \n");

   for(int itera = 1 ; itera <=10 ; itera++){
       extrapolKernel<<<dimGrid, dimBlock>>>(
                                          rs_d,         
                                          extVal_d, phiS_d, jbn_d, d_dPhi,
                                       1.0/deltaX, 1.0/deltaY, 1.0/deltaZ,
                                            Nx, Ny, Nz, 
                                            Flag
                                            );  
     
       RunGK_FirstS<<<dimGrid, dimBlock>>>( d_d, extVal_d, 
                                           dtext, rs_d, Nx, Ny, Nz);
     
       extrapolKernel<<<dimGrid, dimBlock>>>(
                                            rs_d, d_d, 
                                            phiS_d, jbn_d, d_dPhi,
                                       1.0/deltaX, 1.0/deltaY, 1.0/deltaZ,
                                            Nx, Ny, Nz, 
                                            Flag);
     
       RunGK_SecondS<<<dimGrid, dimBlock>>>( d_d, 
                                             extVal_d, d_d, 
                                             dtext, rs_d, 
                                             Nx, Ny, Nz);
     
     
       extrapolKernel<<<dimGrid, dimBlock>>>(
                                            rs_d, d_d, 
                                            phiS_d, jbn_d, d_dPhi,
                                       1.0/deltaX, 1.0/deltaY, 1.0/deltaZ,
                                            Nx, Ny, Nz, 
                                            Flag);
     
       RunGK_ThirdS<<<dimGrid, dimBlock>>>(  extVal_d, 
                                             extVal_d, d_d, 
                                             dtext, rs_d, 
                                             Nx, Ny, Nz);
   }
 // check for error
  hipError_t error = hipGetLastError();
  if(error != hipSuccess)
  {
    // print the CUDA error message and exit
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }

   hipFree(d_d);
   hipFree(d_dPhi);
   hipFree(rs_d);
 
 }
 
 
 
