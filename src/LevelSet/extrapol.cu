#include<hip/hip_runtime.h>
#define MAX(x, y) (((x) > (y)) ? (x) : (y))
#define MIN(x, y) (((x) < (y)) ? (x) : (y))

__global__ void extrapolKernel(
                               double* const rs,          //RS
                               const double* const extVal,//Var extrapol
                               const double* const phiS,  //Level Set F
                               const double* const jbn,   //Jacobian
                               const double* const d_Phi, //Phi Der
                               const double deltaX,
                               const double deltaY,
                               const double deltaZ,
                               const unsigned int Nx,
                               const unsigned int Ny,
                               const unsigned int Nz,
                               const int extFlag
                              )
{

   const int Offset = Nx*Ny*Nz;
   int id2;
   double so;
   double phiDeltaX, phiDeltaY, phiDeltaZ,
         d_ext_xu, d_ext_xd,
         d_ext_yu, d_ext_yd,
         d_ext_zu, d_ext_zd;

   int idx = blockIdx.x*blockDim.x + threadIdx.x,
                      idy = blockIdx.y*blockDim.y + threadIdx.y,
                      idz = blockIdx.z*blockDim.z + threadIdx.z;  


   //Offsets example (id_ip) EQ (i+1,j,k) 
   int id = Nx*Ny*idz + Nx*idy + idx,
                id_ip = Nx*Ny*idz + Nx*idy + idx + 1, 
                id_im = Nx*Ny*idz + Nx*idy + idx - 1, 
                id_jp = Nx*Ny*idz + Nx*(idy + 1) + idx, 
                id_jm = Nx*Ny*idz + Nx*(idy - 1) + idx, 
                id_kp = Nx*Ny*(idz + 1) + Nx*idy + idx, 
                id_km = Nx*Ny*(idz - 1) + Nx*idy + idx; 


   //Dealing with boundaries
   id2 = id;
   if(idx==0){id2 = id_ip; id_im = id;}
   if(idy==0){id2 = id_jp; id_jm = id;} 
   if(idz==0){id2 = id_kp; id_km = id;} 
   if(idx==Nx-1){id2 = id_im ; id_ip = id;} 
   if(idy==Ny-1){id2 = id_jm ; id_jp = id;} 
   if(idz==Nz-1){id2 = id_km ; id_kp = id;} 

  

// pick up the side to extrapol 
   if(extFlag>0){

      so = (double)(phiS[id]>0.0);       
   }
   else{
      so = -1.0*(double)(phiS[id]<=0.0);       
   }

  
   phiDeltaX = so*d_Phi[id           ];
   phiDeltaY = so*d_Phi[id + 1*Offset];
   phiDeltaZ = so*d_Phi[id + 2*Offset];


// Downwind derivatives of ext 
   d_ext_xd = deltaX*jbn[id           ]*(extVal[id2] - extVal[id_im]); 
   d_ext_yd = deltaY*jbn[id + 4*Offset]*(extVal[id2] - extVal[id_jm]); 
   d_ext_zd = deltaZ*jbn[id + 8*Offset]*(extVal[id2] - extVal[id_km]); 

// Upwind derivatives of ext 
   d_ext_xu = deltaX*jbn[id           ]*(extVal[id_ip] - extVal[id2]); 
   d_ext_yu = deltaY*jbn[id + 4*Offset]*(extVal[id_jp] - extVal[id2]); 
   d_ext_zu = deltaZ*jbn[id + 8*Offset]*(extVal[id_kp] - extVal[id2]); 
   

   double xMax = (double)(phiDeltaX > 0.0) 
               - (double)(phiDeltaX < 0.0);

   rs[id] = (0.5*(xMax + 1.0)*d_ext_xd 
          + 0.5*abs(xMax - 1.0)*d_ext_xu)*phiDeltaX;

   xMax = (double)(phiDeltaY > 0.0) 
        - (double)(phiDeltaY < 0.0);

   rs[id] += (0.5*(xMax + 1.0)*d_ext_yd 
          + 0.5*abs(xMax - 1.0)*d_ext_yu)*phiDeltaY;


   xMax = (double)(phiDeltaZ > 0.0) 
        - (double)(phiDeltaZ < 0.0);

   rs[id] += (0.5*(xMax + 1.0)*d_ext_zd 
          + 0.5*abs(xMax - 1.0)*d_ext_zu)*phiDeltaZ;

          
   return; 

}

__global__ void DevFirstOrder_LS(
                                double* const d_Phi,
                                const double* const phiS,
                                const double* const jbn,
                                const double deltaX,
                                const double deltaY,
                                const double deltaZ,
                                const unsigned int Nx, 
                                const unsigned int Ny, 
                                const unsigned int Nz
                                )
{

   const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x,
                      idy = blockIdx.y*blockDim.y + threadIdx.y,
                      idz = blockIdx.z*blockDim.z + threadIdx.z;  

   //Offsets example (id_ip) EQ (i+1,j,k) 
   unsigned int id = Nx*Ny*idz + Nx*idy + idx,
                id_ip = Nx*Ny*idz + Nx*idy + idx + 1, 
                id_im = Nx*Ny*idz + Nx*idy + idx - 1, 
                id_jp = Nx*Ny*idz + Nx*(idy + 1) + idx, 
                id_jm = Nx*Ny*idz + Nx*(idy - 1) + idx, 
                id_kp = Nx*Ny*(idz + 1) + Nx*idy + idx, 
                id_km = Nx*Ny*(idz - 1) + Nx*idy + idx; 
   
   double factor = 0.5;

   //Dealing with boundaries
   if(idx==0){id_im = id; factor = 1.0;}
   if(idy==0){id_jm = id; factor = 1.0;}
   if(idz==0){id_km = id; factor = 1.0;}
   if(idx==Nx-1){id_ip = id; factor = 1.0;}
   if(idy==Ny-1){id_jp = id; factor = 1.0;}
   if(idz==Nz-1){id_kp = id; factor = 1.0;}

   const unsigned int Offset = Nx*Ny*Nz;

   d_Phi[           id] = factor*deltaX*jbn[id           ]
                        * (phiS[id_ip] - phiS[id_im]);

   d_Phi[1*Offset + id] = factor*deltaY*jbn[id + 4*Offset]
                        * (phiS[id_jp] - phiS[id_jm]);

   d_Phi[2*Offset + id] = factor*deltaZ*jbn[id + 8*Offset]
                        * (phiS[id_kp] - phiS[id_km]);
	
   return;
}

__global__ void RunGK_FirstS(
                            double* d,
                            double* d0,
                            double  dt, 
                            double* rs,
                            const int Nx, const int Ny, const int Nz
                            )
{
   const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x,
                      idy = blockIdx.y*blockDim.y + threadIdx.y,
                      idz = blockIdx.z*blockDim.z + threadIdx.z;

   const unsigned int id = idx + idy*Nx + idz*Nx*Ny;
   
   d[id] = d0[id] - dt*rs[id];

   return;
}

__global__ void RunGK_SecondS(
                             double* d,
                             double* d0,
                             double* d1,
                             double  dt, 
                             double* rs,
                             const int Nx, const int Ny, const int Nz
                            )
{
   const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x,
                      idy = blockIdx.y*blockDim.y + threadIdx.y,
                      idz = blockIdx.z*blockDim.z + threadIdx.z;

   const unsigned int id = idx + idy*Nx + idz*Nx*Ny;
   
   d[id] = 0.75*d0[id] +0.25*( d1[id] - dt*rs[id]);

   return;
}

__global__ void RunGK_ThirdS(
                             double* d,
                             double* d0,
                             double* d1,
                             const double dt, 
                             double* rs,
                             const int Nx, const int Ny, const int Nz
                            )
{
   const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x,
                      idy = blockIdx.y*blockDim.y + threadIdx.y,
                      idz = blockIdx.z*blockDim.z + threadIdx.z;

   const unsigned int id = idx + idy*Nx + idz*Nx*Ny;
   
   d[id] = (d0[id] + 2.0*( d1[id] - dt*rs[id])) / 3.0 ;

   return;
}

__global__ void copyLSGas(
                          double* const value,
                          const double* const copyVal, 
                          const double* const phiS,
                          int Nx, int Ny, int Nz
                         )
{

   const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x,
                      idy = blockIdx.y*blockDim.y + threadIdx.y,
                      idz = blockIdx.z*blockDim.z + threadIdx.z;

   const unsigned int id = idx + idy*Nx + idz*Nx*Ny;
   
   value[id] = (phiS[id] > 0.0) ? copyVal[id] : value[id];
   
   return;
}


__global__ void copyLSLiquid(
                             double* const value,
                             const double* const copyVal, 
                             const double* const phiS,
                             int Nx, int Ny, int Nz,
                             int disp
                            )
{

   const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x,
                      idy = blockIdx.y*blockDim.y + threadIdx.y,
                      idz = blockIdx.z*blockDim.z + threadIdx.z;

   const unsigned int id = idx + idy*Nx + idz*Nx*Ny;

   
   const unsigned int offset = Nx*Ny*Nz;
   
   value[id + disp*offset] = (phiS[id] < 0.0) ? copyVal[id] :  
                                     value[id + disp*offset];
   
   return;
}
