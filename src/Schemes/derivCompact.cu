#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>
#include"Engine/cuArray.hpp"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

template<typename Tprec>
__global__
void transposeArrayJ(ArrayDev3D<Tprec> a, ArrayDev3D<Tprec> b){

   int i = blockDim.x*blockIdx.x + threadIdx.x;
   int j = blockDim.y*blockIdx.y + threadIdx.y;
   int k = blockDim.z*blockIdx.z + threadIdx.z;

   if(i < a.getDim(1) && j < a.getDim(2) && k < a.getDim(3)){

      a(i,j,k) = b(j,i,k);

   }

}

template<typename Tprec>
__global__
void transposeArrayK(ArrayDev3D<Tprec> a, ArrayDev3D<Tprec> b){

   int i = blockDim.x*blockIdx.x + threadIdx.x;
   int j = blockDim.y*blockIdx.y + threadIdx.y;
   int k = blockDim.z*blockIdx.z + threadIdx.z;

   if(i < a.getDim(1) && j < a.getDim(2) && k < a.getDim(3)){

      a(i,j,k) = b(k,j,i);

   }
}



template<typename Tprec>
__global__
void getC4RHS(ArrayDev3D<Tprec> f,  ArrayDev3D<Tprec> r, Tprec delta){

   int idx = blockDim.x*blockIdx.x + threadIdx.x,
       idy = blockDim.y*blockIdx.y + threadIdx.y,
       idz = blockDim.z*blockIdx.z + threadIdx.z;

   Tprec ars  =   3.0/4.0,
         ars1 = -17.0/6.0,
         brs1 =   3.0/2.0,
         crs1 =   3.0/2.0,
         drs1 =  -1.0/6.0;

   if(idx > 0 && idx < r.getDim(1) - 1 && idy < r.getDim(2) && 
      idz < r.getDim(3)){
     
      f(idx,idy,idz) =   (1.0/delta)*(ars*(r(idx+1,idy,idz) 
                                          -r(idx-1,idy,idz))); 
   }

   if(idx == 0 && idy < r.getDim(2) && idz < r.getDim(3)){
        f(idx,idy,idz) = (1.0/delta)*(ars1*r(0,idy,idz)
                                     +brs1*r(1,idy,idz)
                                     +crs1*r(2,idy,idz) 
                                     +drs1*r(3,idy,idz)
                                     );
   }

   if(idx == r.getDim(1) - 1 && idy < r.getDim(2) && idz < r.getDim(3)){
        f(idx,idy,idz) = (1.0/delta)*(-ars1*r(r.getDim(1) - 1,idy,idz)
                                      -brs1*r(r.getDim(1) - 2,idy,idz)
                                      -crs1*r(r.getDim(1) - 3,idy,idz) 
                                      -drs1*r(r.getDim(1) - 4,idy,idz)
                                     );
   }

}


template<typename Tprec>
__global__
void getC6RHS(ArrayDev3D<Tprec> f,  ArrayDev3D<Tprec> r, Tprec delta){

   int idx = blockDim.x*blockIdx.x + threadIdx.x,
       idy = blockDim.y*blockIdx.y + threadIdx.y,
       idz = blockDim.z*blockIdx.z + threadIdx.z;


   Tprec ars  =   14.0/(9.0*2.0),
         brs  =   1.0/(9.0*4.0),
         ars1 = -17.0/6.0,
         brs1 =   3.0/2.0,
         crs1 =   3.0/2.0,
         drs1 =  -1.0/6.0,
         ars2 =  -3.0/4.0,
         brs2 =       0.0,
         crs2 =   3.0/4.0,
         drs2 =       0.0;


   if(idx > 1 && idx < r.getDim(1) - 2 && idy < r.getDim(2) && 
      idz < r.getDim(3)){
     
      f(idx,idy,idz) =   (1.0/delta)*(ars*(r(idx+1,idy,idz)-r(idx-1,idy,idz)) 
                          + (brs*(r(idx+2,idy,idz)-r(idx-2,idy,idz)))); 
   }

   if(idx == 0 && idy < r.getDim(2) && idz < r.getDim(3)){
        f(idx,idy,idz) = (1.0/delta)*(ars1*r(0,idy,idz)
                                     +brs1*r(1,idy,idz)
                                     +crs1*r(2,idy,idz) 
                                     +drs1*r(3,idy,idz)
                                     );
   }


   if(idx == 1 && idy < r.getDim(2) && idz < r.getDim(3)){
        f(idx,idy,idz) = (1.0/delta)*(ars2*r(0,idy,idz)
                                     +brs2*r(1,idy,idz)
                                     +crs2*r(2,idy,idz) 
                                     +drs2*r(3,idy,idz)
                                     );
   }

   if(idx == r.getDim(1) - 1 && idy < r.getDim(2) && idz < r.getDim(3)){
        f(idx,idy,idz) = (1.0/delta)*(-ars1*r(r.getDim(1) - 1,idy,idz)
                                      -brs1*r(r.getDim(1) - 2,idy,idz)
                                      -crs1*r(r.getDim(1) - 3,idy,idz) 
                                      -drs1*r(r.getDim(1) - 4,idy,idz)
                                     );
   }

   if(idx == r.getDim(1) - 2 && idy < r.getDim(2) && idz < r.getDim(3)){
        f(idx,idy,idz) = (1.0/delta)*(-ars2*r(r.getDim(1) - 1,idy,idz)
                                      -brs2*r(r.getDim(1) - 2,idy,idz)
                                      -crs2*r(r.getDim(1) - 3,idy,idz) 
                                      -drs2*r(r.getDim(1) - 4,idy,idz)
                                     );
   }

}



template<typename Tprec>
__global__
void cuDerivCS(ArrayDev3D<Tprec> f, 
             ArrayDev3D<Tprec> r,
             ArrayDev3D<Tprec> gam
            ){

   extern __shared__ Tprec diag[]; 

   int block     = blockDim.x*blockDim.y;
   int blockloop = 3*f.getDim(1) / block; 

   int id_inside;
   
   if(3*f.getDim(1) % block) blockloop += 1;

   for(int it = 0; it < blockloop; it++){
     id_inside = it*block + blockDim.x*threadIdx.y + threadIdx.x;

     //Diagonal a matrix
     if(id_inside < f.getDim(1)){
       diag[id_inside] = 1.0/3.0; 
     } 
     //Diagonal b matrix
     if(id_inside >= f.getDim(1) && id_inside < 2*f.getDim(1)){
       diag[id_inside] = 1.0; 
     } 

     //Diagonal c matrix
     if(id_inside >= 2*f.getDim(1) && id_inside < 3*f.getDim(1)){
       diag[id_inside] = 1.0/3.0; 
     } 

//   A cof
     if(id_inside == 0            ) diag[id_inside] = 0.0;
     if(id_inside == 1            ) diag[id_inside] = 1.0 / 4.0;
     if(id_inside == f.getDim(1) - 1) diag[id_inside] = 3.0;
     if(id_inside == f.getDim(1) - 2) diag[id_inside] = 1.0/4.0;

//   C cof
     if(id_inside == 2*f.getDim(1)    ) diag[id_inside] = 3.0;                
     if(id_inside == 2*f.getDim(1) + 1) diag[id_inside] = 1.0/4.0;
     if(id_inside == 3*f.getDim(1) - 1) diag[id_inside] = 0.0;
     if(id_inside == 3*f.getDim(1) - 2) diag[id_inside] = 1.0/4.0;
   }
   __syncthreads();


   int idx = blockDim.x*blockIdx.x + threadIdx.x;  
   int idy = blockDim.y*blockIdx.y + threadIdx.y;  

   if(idx < f.getDim(2) && idy < f.getDim(3)){

      Tprec bet = diag[f.getDim(1)];
    
      f(0,idx,idy) = r(0,idx,idy) / bet; 
      for(int i = 1; i <f.getDim(1); i++){  
           gam(i,idx,idy) = diag[2*f.getDim(1) + i -1] / bet;
           bet = diag[f.getDim(1) + i] - diag[i]*gam(i,idx,idy);
    
         f(i,idx,idy) =  (r(i,idx,idy) - diag[i]*f(i-1,idx,idy))/bet;
      }
    
      for(int i = f.getDim(1) - 2; i >= 0 ; i--){  
         f(i,idx,idy) = f(i,idx,idy) - gam(i+1,idx,idy)*f(i+1,idx,idy);
    
      }

   }
}


//Deriv functions
namespace cu{
namespace cs{

template<typename Tprec>
void dFdX(
            ArrayDev3D<Tprec>  &fprima, 
            ArrayDev3D<Tprec>  &f, 
            Tprec              delta
           ){

    Tprec* rhs_d;

    hipMalloc((void **)&rhs_d, f.getDim(1)*f.getDim(2)*f.getDim(3)*sizeof(Tprec));

    ArrayDev3D<Tprec> rhs(f.getDim(1), f.getDim(2), f.getDim(3)),
                      trash(f.getDim(1), f.getDim(2), f.getDim(3));


//--configuring launch options (fermi card)


    int sharedSize = 3*rhs.getDim(1)*sizeof(Tprec);

    int gridDimX, gridDimY, gridDimZ;
    int gridDimYDer, gridDimZDer;
   
    gridDimX = f.getDim(1) / 8; 
    if(f.getDim(1) % 8) gridDimX++;

    gridDimY = f.getDim(2) / 8; 
    if(f.getDim(2) % 8) gridDimY++;

    gridDimYDer = f.getDim(2) / 16; 
    if(f.getDim(2) % 16) gridDimYDer++;

    gridDimZ = f.getDim(3) / 4; 
    if(f.getDim(3) % 4) gridDimZ++;

    gridDimZDer = f.getDim(3) / 16; 
    if(f.getDim(3) % 16) gridDimZDer++;

    dim3 BlockDim(8,8,4);
    dim3 GridDim(gridDimX,gridDimY,gridDimZ);

    dim3 BlockDimDer(16,16);
    dim3 GridDimDer(gridDimYDer,gridDimZDer);


//--end configuring launch options (fermi card)

      getC6RHS<<<GridDim,BlockDim>>>(rhs, f, delta);
      cuDerivCS<<<GridDimDer,BlockDimDer,sharedSize>>>(fprima,
                                                       rhs, 
                                                       trash);

     hipError_t error = hipGetLastError();
     if(error != hipSuccess)
     {   
        // print the CUDA error message and exit
        printf("CUDA dFdX error: %s\n", hipGetErrorString(error));
        exit(-1);
     }  

     hipFree(rhs_d);
 
}

template<typename Tprec>
void dFdY(
            ArrayDev3D<Tprec>  &fprima, 
            ArrayDev3D<Tprec>  &f, 
            Tprec               delta
           ){

    ArrayDev3D<Tprec> rhs(f.getDim(2), f.getDim(1), f.getDim(3)),
                      ftrans(f.getDim(2), f.getDim(1), f.getDim(3)),
                      trash(f.getDim(2), f.getDim(1), f.getDim(3));
    
//--configuring launch options (fermi card)
   
    int gridDimX, gridDimY, gridDimZ;
    int gridDimYDer, gridDimZDer;

    int sharedSize = 3*rhs.getDim(1)*sizeof(Tprec);

    gridDimX = ftrans.getDim(1) / 8; 
    if(ftrans.getDim(1) % 8) gridDimX++;

    gridDimY = ftrans.getDim(2) / 8; 
    if(ftrans.getDim(2) % 8) gridDimY++;

    gridDimYDer = ftrans.getDim(2) / 16; 
    if(ftrans.getDim(2) % 16) gridDimYDer++;

    gridDimZ = ftrans.getDim(3) / 4; 
    if(ftrans.getDim(3) % 4) gridDimZ++;

    gridDimZDer = ftrans.getDim(3) / 16; 
    if(ftrans.getDim(3) % 16) gridDimZDer++;

    dim3 BlockDim(8,8,4);
    dim3 GridDim(gridDimX,gridDimY,gridDimZ);

    dim3 BlockDimDer(16,16);
    dim3 GridDimDer(gridDimYDer,gridDimZDer);

//--end configuation

    transposeArrayJ<<<GridDim,BlockDim>>>(ftrans,f);

    getC6RHS<<<GridDim,BlockDim>>>(rhs, ftrans, delta);
    cuDerivCS<<<GridDimDer,BlockDimDer,sharedSize>>>(ftrans,
                                                     rhs, 
                                                     trash);

    GridDim = dim3(gridDimY,gridDimX,gridDimZ);
    transposeArrayJ<<<GridDim,BlockDim>>>(fprima, ftrans);

     hipError_t error = hipGetLastError();
     if(error != hipSuccess)
     {   
        // print the CUDA error message and exit
        printf("CUDA dFdY error: %s\n", hipGetErrorString(error));
        exit(-1);
     }  

}

template<typename Tprec>
void dFdZ(
          ArrayDev3D<Tprec>  &fprima, 
          ArrayDev3D<Tprec>  &f, 
          Tprec               delta
          ){

    ArrayDev3D<Tprec> rhs(f.getDim(3), f.getDim(2), f.getDim(1)),
                      ftrans(f.getDim(3), f.getDim(2), f.getDim(1)),
                      trash(f.getDim(3), f.getDim(2), f.getDim(1));
    
//--configuring launch options (fermi card)

   
    int gridDimX, gridDimY, gridDimZ;
    int gridDimYDer, gridDimZDer;

    int sharedSize = 3*rhs.getDim(1)*sizeof(Tprec);

    gridDimX = ftrans.getDim(1) / 8; 
    if(ftrans.getDim(1) % 8) gridDimX++;

    gridDimY = ftrans.getDim(2) / 8; 
    if(ftrans.getDim(2) % 8) gridDimY++;

    gridDimYDer = ftrans.getDim(2) / 16; 
    if(ftrans.getDim(2) % 16) gridDimYDer++;

    gridDimZ = ftrans.getDim(3) / 4; 
    if(ftrans.getDim(3) % 4) gridDimZ++;

    gridDimZDer = ftrans.getDim(3) / 16; 
    if(ftrans.getDim(3) % 16) gridDimZDer++;

    dim3 BlockDim(8,8,4);
    dim3 GridDim(gridDimX,gridDimY,gridDimZ);

    dim3 BlockDimDer(16,16);
    dim3 GridDimDer(gridDimYDer,gridDimZDer);

//--end configuation

    transposeArrayK<<<GridDim,BlockDim>>>(ftrans,f);

    getC6RHS<<<GridDim,BlockDim>>>(rhs, ftrans, delta);
    cuDerivCS<<<GridDimDer,BlockDimDer,sharedSize>>>(ftrans,
                                                     rhs, 
                                                     trash);

    BlockDim = dim3(4,8,8);
    GridDim  = dim3(gridDimZ,gridDimY,gridDimX);
    transposeArrayK<<<GridDim,BlockDim>>>(fprima, ftrans);

     hipError_t error = hipGetLastError();
     if(error != hipSuccess)
     {   
        // print the CUDA error message and exit
        printf("CUDA dFdZ error: %s\n", hipGetErrorString(error));
        exit(-1);
     }  


}

//end namespaces
}
}

