#include<hip/hip_runtime.h>

//todo 

template<typename Tprec>
__host__ 
ArrayDev<Tprec>::ArrayDev(int _dimsize){

  dimsize = _dimsize > 0? _dimsize: 1;

  allocate();

  isCopy = false;

}

template<typename Tprec>
ArrayDev<Tprec>::ArrayDev(ArrayDev<Tprec> &_orig){

  *this = _orig;
  isCopy = true;

}


template<typename Tprec>
ArrayDev<Tprec>::~ArrayDev(){
     if(!isCopy){
       hipFree(d_data);
     }

}

    
template<typename Tprec>
__host__
void ArrayDev<Tprec>::resize(int _dimsize){


      dimsize = _dimsize > 0? _dimsize: 1;
      allocate();
      isCopy = false;
}


template<typename Tprec>
__device__ 
Tprec& ArrayDev<Tprec>::operator()(int i) 
{

      return  d_data[i];
}


/*
template<typename Tprec>
__host__
ArrayDev<Tprec>& ArrayDev<Tprec>::operator=(ArrayDev<Tprec>& array){

     if(this != &array){
        dimsize = array.getDim();
        allocate();

        copyDevToDev(array.devPtr());
     }

   return *this;
}
*/

template<typename Tprec>
__host__ __device__ 
int ArrayDev<Tprec>::getDim() const { 
    return dimsize;
} 


template<typename Tprec>
__host__ 
Tprec* ArrayDev<Tprec>::devPtr(){
    return d_data; 
} 

template<typename Tprec>
__host__ void ArrayDev<Tprec>::allocate(){

  hipError_t retult = 
  hipMalloc((void**)&d_data, dimsize*sizeof(Tprec));
}


template<typename Tprec>
__host__ 
void ArrayDev<Tprec>::copyToHost(Tprec* hostPtr) {

   hipMemcpy(hostPtr, d_data, dimsize*sizeof(Tprec), 
              hipMemcpyDeviceToHost);
   
}

template<typename Tprec>
__host__ 
void ArrayDev<Tprec>::copyFromHost(Tprec* hostPtr){

   hipMemcpy(d_data, hostPtr, dimsize*sizeof(Tprec), 
              hipMemcpyHostToDevice);
   
}

template<typename Tprec>
__host__ 
void ArrayDev<Tprec>::copyDevToDev(Tprec* devPtr){

   hipMemcpy(d_data, devPtr, dimsize*sizeof(Tprec), 
              hipMemcpyDeviceToDevice);
   
}

template<typename Tprec>
__device__ 
Tprec& ArrayDev3D<Tprec>::operator()(int i, int j, int k) 
{
      return  ArrayDev<Tprec>::d_data[nx*ny*k + nx*j + i];
}


