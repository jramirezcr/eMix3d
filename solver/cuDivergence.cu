#include "hip/hip_runtime.h"
#include"Schemes/derivCompact.hpp"
#include"Engine/cuArray.hpp"

template<typename Tprec>
__global__ void kernProyectDataJBN(
                          ArrayDev3D<Tprec> FLUX,
                          ArrayDev3D<Tprec> jbndet,
                          ArrayDev3D<Tprec> jbn
                          ){


   int i = blockIdx.x*blockDim.x + threadIdx.x;
   int j = blockIdx.y*blockDim.y + threadIdx.y;
   int k = blockIdx.z*blockDim.z + threadIdx.z;

   if(i < FLUX.getDim(1) && j < FLUX.getDim(2) && k < FLUX.getDim(3)){
       FLUX(i,j,k) = jbndet(i,j,k)*jbn(i,j,k)*FLUX(i,j,k);
   }
}

template<typename Tprec>
__global__ void kernGetDiv(
                          ArrayDev3D<Tprec> DIV,
                          ArrayDev3D<Tprec> jbn,
                          ArrayDev3D<Tprec> FLUXE,
                          ArrayDev3D<Tprec> FLUXF,
                          ArrayDev3D<Tprec> FLUXG
                          ){

   int i = blockIdx.x*blockDim.x + threadIdx.x;
   int j = blockIdx.y*blockDim.y + threadIdx.y;
   int k = blockIdx.z*blockDim.z + threadIdx.z;

   if(i < DIV.getDim(1) && j < DIV.getDim(2) && k < DIV.getDim(3)){
       DIV(i,j,k) = (FLUXE(i,j,k) + FLUXF(i,j,k) + FLUXG(i,j,k))*
                    jbn(i,j,k);
   }
}

template<typename Tprec>
void getDiv(
            ArrayDev3D<Tprec> DIV,
            ArrayDev3D<Tprec> jbn,
            ArrayDev3D<Tprec> FLUXE,
            ArrayDev3D<Tprec> FLUXF,
            ArrayDev3D<Tprec> FLUXG
            ){

    int gridDimX, gridDimY, gridDimZ;
     
    gridDimX = DIV.getDim(1) / 8; 
    if(DIV.getDim(1) % 8) gridDimX++;

    gridDimY = DIV.getDim(2) / 8; 
    if(DIV.getDim(2) % 8) gridDimY++;

    gridDimZ = DIV.getDim(3) / 4; 
    if(DIV.getDim(3) % 4) gridDimZ++;

    dim3 BlockDim(8,8,4);
    dim3 GridDim(gridDimX,gridDimY,gridDimZ);

    kernGetDiv<<<GridDim,BlockDim>>>(DIV,jbn,FLUXE,FLUXF,FLUXG);

    hipError_t error; 
    error = hipGetLastError();
    if(error!=hipSuccess){
      std::cout<< "cuda error getDiv: " << hipGetErrorString(error) <<std::endl;
      exit(-1);
    
    }
}

template<typename Tprec>
void proyectDataJBN(
                    ArrayDev3D<Tprec> FLUX,
                    ArrayDev3D<Tprec> jbndet,
                    ArrayDev3D<Tprec> jbn
                   ){
    int gridDimX, gridDimY, gridDimZ;
     
    gridDimX = FLUX.getDim(1) / 8; 
    if(FLUX.getDim(1) % 8) gridDimX++;

    gridDimY = FLUX.getDim(2) / 8; 
    if(FLUX.getDim(2) % 8) gridDimY++;

    gridDimZ = FLUX.getDim(3) / 4; 
    if(FLUX.getDim(3) % 4) gridDimZ++;

    dim3 BlockDim(8,8,4);
    dim3 GridDim(gridDimX,gridDimY,gridDimZ);

    kernProyectDataJBN<<<GridDim, BlockDim>>>(FLUX, jbndet, jbn);

    hipError_t error; 
    error = hipGetLastError();
    if(error!=hipSuccess){
      std::cout<< "cuda error proyectDataJBN: " << hipGetErrorString(error) <<std::endl;
      exit(-1);
    }

}

extern "C"{

void cuDivergence(
                 double *rs,
                 double *rsv,
                 double *e,
                 double *f, 
                 double *g,
                 double *ev,
                 double *fv, 
                 double *gv,
                 double *jbn,
                 double deltaX, double deltaY, double deltaZ,
                 int Nx, int Ny, int Nz 
                 ){

   int offset = Nx*Ny*Nz; 


   ArrayDev3D<double> FLUXE(Nx,Ny,Nz);
   ArrayDev3D<double> FLUXF(Nx,Ny,Nz);
   ArrayDev3D<double> FLUXG(Nx,Ny,Nz);

   ArrayDev3D<double> FLUXEv(Nx,Ny,Nz);
   ArrayDev3D<double> FLUXFv(Nx,Ny,Nz);
   ArrayDev3D<double> FLUXGv(Nx,Ny,Nz);

   ArrayDev3D<double> dEdx(Nx,Ny,Nz);
   ArrayDev3D<double> dFdy(Nx,Ny,Nz);
   ArrayDev3D<double> dGdz(Nx,Ny,Nz);

   ArrayDev3D<double> dEvdx(Nx,Ny,Nz);
   ArrayDev3D<double> dFvdy(Nx,Ny,Nz);
   ArrayDev3D<double> dGvdz(Nx,Ny,Nz);

   ArrayDev3D<double> jbn0(Nx,Ny,Nz);
   ArrayDev3D<double> jbn4(Nx,Ny,Nz);
   ArrayDev3D<double> jbn8(Nx,Ny,Nz);

   ArrayDev3D<double> jbndet(Nx,Ny,Nz);
   ArrayDev3D<double> jbn10(Nx,Ny,Nz);

   ArrayDev3D<double> rhs(Nx,Ny,Nz);
   ArrayDev3D<double> rhsv(Nx,Ny,Nz);

   FLUXE.copyFromHost(e);
   FLUXF.copyFromHost(f);
   FLUXG.copyFromHost(g);

   FLUXEv.copyFromHost(ev);
   FLUXFv.copyFromHost(fv);
   FLUXGv.copyFromHost(gv);

   jbn0.copyFromHost(&(jbn[0*offset]));
   jbn4.copyFromHost(&(jbn[4*offset]));
   jbn8.copyFromHost(&(jbn[8*offset]));

   jbndet.copyFromHost(&(jbn[9*offset]));
   jbn10.copyFromHost(&(jbn[10*offset]));


   proyectDataJBN(FLUXE, jbndet, jbn0);
   proyectDataJBN(FLUXF, jbndet, jbn4);
   proyectDataJBN(FLUXG, jbndet, jbn8);

   proyectDataJBN(FLUXEv, jbndet, jbn0);
   proyectDataJBN(FLUXFv, jbndet, jbn4);
   proyectDataJBN(FLUXGv, jbndet, jbn8);

   cu::cs::dFdX(dEdx,  FLUXE ,  deltaX);
   cu::cs::dFdX(dEvdx, FLUXEv , deltaX);

   cu::cs::dFdY(dFdy,  FLUXF ,  deltaY);
   cu::cs::dFdY(dFvdy, FLUXFv , deltaY);

   cu::cs::dFdZ(dGdz,  FLUXG ,  deltaZ);
   cu::cs::dFdZ(dGvdz, FLUXGv , deltaZ);

   getDiv(rhs, jbn10, dEdx, dFdy, dGdz);
   getDiv(rhsv, jbn10, dEvdx, dFvdy, dGvdz);

   rhs.copyToHost(rs);
   rhsv.copyToHost(rsv);

}

}
