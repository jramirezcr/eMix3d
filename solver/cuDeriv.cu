#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>

#include"Schemes/derivCompact.hpp"
#include"Engine/cuArray.hpp"

template<typename Tprec>
struct TensorDev2{

   TensorDev2(int i, int j, int k){
      resize(i,j,k);
   }

   void resize(int i, int j, int k){
       e11.resize(i,j,k);
       e21.resize(i,j,k);
       e31.resize(i,j,k);
       e12.resize(i,j,k);
       e22.resize(i,j,k);
       e32.resize(i,j,k);
       e13.resize(i,j,k);
       e23.resize(i,j,k);
       e33.resize(i,j,k);
    
   }

   ArrayDev3D<Tprec> e11, e21, e31; 
   ArrayDev3D<Tprec> e12, e22, e32; 
   ArrayDev3D<Tprec> e13, e23, e33; 

};

template<typename Tprec>
struct PseudoConservative{

   PseudoConservative(int i, int j, int k){
      resize(i,j,k);
   }

   void resize(int i, int j, int k){
       press.resize(i,j,k);
       rhoU.resize(i,j,k);
       rhoV.resize(i,j,k);
       rhoW.resize(i,j,k);
   }

   ArrayDev3D<Tprec> press, rhoU, rhoV, rhoW; 
};


template<typename Tprec>
struct Fluxes{

   Fluxes(int i, int j, int k){
      resize(i,j,k);
   }

   void resize(int i, int j, int k){
       e.resize(i,j,k);
       f.resize(i,j,k);
       g.resize(i,j,k);

       ev.resize(i,j,k);
       fv.resize(i,j,k);
       gv.resize(i,j,k);
   }

   ArrayDev3D<Tprec> e, f, g; 
   ArrayDev3D<Tprec> ev, fv, gv; 
};



template<typename Tprec>
struct VectorDev3D{

   VectorDev3D(int i, int j, int k){
      resize(i,j,k);
   }

   void resize(int i, int j, int k){
       x.resize(i,j,k);
       y.resize(i,j,k);
       z.resize(i,j,k);

   }

   ArrayDev3D<Tprec> x, y, z; 
};



template<typename Tprec>
struct FlowVariables{

   Tprec coefMa; 
   Tprec coefRe; 
   Tprec coefPr; 
   Tprec coefFr; 

};


template<typename Tprec>
__global__
void cuGetGradientJacob(TensorDev2<Tprec> Tensor,
                        ArrayDev3D<Tprec> jbnX,
                        ArrayDev3D<Tprec> jbnY,
                        ArrayDev3D<Tprec> jbnZ
                        ){

     int i = blockDim.x*blockIdx.x + threadIdx.x;
     int j = blockDim.y*blockIdx.y + threadIdx.y;
     int k = blockDim.z*blockIdx.z + threadIdx.z;

     if(i < Tensor.e11.getDim(1) && j < Tensor.e11.getDim(2) 
                                 && k < Tensor.e11.getDim(3)){

        Tensor.e11(i,j,k) = Tensor.e11(i,j,k)*jbnX(i,j,k);
        Tensor.e12(i,j,k) = Tensor.e12(i,j,k)*jbnX(i,j,k);
        Tensor.e13(i,j,k) = Tensor.e13(i,j,k)*jbnX(i,j,k);

        Tensor.e21(i,j,k) = Tensor.e21(i,j,k)*jbnY(i,j,k);
        Tensor.e22(i,j,k) = Tensor.e22(i,j,k)*jbnY(i,j,k);
        Tensor.e23(i,j,k) = Tensor.e23(i,j,k)*jbnY(i,j,k);

        Tensor.e31(i,j,k) = Tensor.e31(i,j,k)*jbnZ(i,j,k);
        Tensor.e32(i,j,k) = Tensor.e32(i,j,k)*jbnZ(i,j,k);
        Tensor.e33(i,j,k) = Tensor.e33(i,j,k)*jbnZ(i,j,k);
     }
}

template<typename Tprec>
__global__
void cuGetViscousSGWALLE(ArrayDev3D<Tprec> vt, 
                         ArrayDev3D<Tprec> Delta, 
                         ArrayDev3D<Tprec> SdSd, 
                         ArrayDev3D<Tprec> SS
                         ){

     int i = blockDim.x*blockIdx.x + threadIdx.x;
     int j = blockDim.y*blockIdx.y + threadIdx.y;
     int k = blockDim.z*blockIdx.z + threadIdx.z;

     if(i < SdSd.getDim(1) && j < SdSd.getDim(2) 
                             && k < SdSd.getDim(3)){

       vt(i,j,k)  = pow(0.5*Delta(i,j,k),2.0)*pow(SdSd(i,j,k),(3.0/2.0))  ;
       vt(i,j,k) /=(pow(SS(i,j,k),(5.0/2.0)) + pow(SdSd(i,j,k),(5.0/4.0)) + 
                    0.00000000000000001);
     }
}

template<typename Tprec>
__global__
void cuGetSdTensorWALLE(TensorDev2<Tprec> Sd,
                        TensorDev2<Tprec> S2,
                        TensorDev2<Tprec> R2,
                        ArrayDev3D<Tprec> SS, 
                        ArrayDev3D<Tprec> RR
                        ){

     int i = blockDim.x*blockIdx.x + threadIdx.x;
     int j = blockDim.y*blockIdx.y + threadIdx.y;
     int k = blockDim.z*blockIdx.z + threadIdx.z;

     if(i < Sd.e11.getDim(1) && j < Sd.e11.getDim(2) 
                             && k < Sd.e11.getDim(3)){

          Sd.e11(i,j,k) = S2.e11(i,j,k) + R2.e11(i,j,k) -
                          (SS(i,j,k) + RR(i,j,k))/3.0;

          Sd.e22(i,j,k) = S2.e22(i,j,k) + R2.e22(i,j,k) -
                          (SS(i,j,k) + RR(i,j,k))/3.0;

          Sd.e33(i,j,k) = S2.e33(i,j,k) + R2.e33(i,j,k) -
                          (SS(i,j,k) + RR(i,j,k))/3.0;

          Sd.e12(i,j,k) = S2.e12(i,j,k) + R2.e12(i,j,k);

          Sd.e13(i,j,k) = S2.e13(i,j,k) + R2.e13(i,j,k);

          Sd.e21(i,j,k) = S2.e21(i,j,k) + R2.e21(i,j,k);

          Sd.e23(i,j,k) = S2.e23(i,j,k) + R2.e23(i,j,k);

          Sd.e31(i,j,k) = S2.e31(i,j,k) + R2.e31(i,j,k);

          Sd.e32(i,j,k) = S2.e32(i,j,k) + R2.e32(i,j,k);

     }
}

template<typename Tprec>
__global__
void cuGetDoubleContraction(ArrayDev3D<Tprec> result, 
                           TensorDev2<Tprec>  Tensor
                           ){

     int i = blockDim.x*blockIdx.x + threadIdx.x;
     int j = blockDim.y*blockIdx.y + threadIdx.y;
     int k = blockDim.z*blockIdx.z + threadIdx.z;

     if(i < Tensor.e11.getDim(1) && j < Tensor.e11.getDim(2) 
                                 && k < Tensor.e11.getDim(3)){

        result(i,j,k) = Tensor.e11(i,j,k)*Tensor.e11(i,j,k) +
                        Tensor.e22(i,j,k)*Tensor.e22(i,j,k) +
                        Tensor.e33(i,j,k)*Tensor.e33(i,j,k) +
                        Tensor.e12(i,j,k)*Tensor.e12(i,j,k) +
                        Tensor.e13(i,j,k)*Tensor.e13(i,j,k) +
                        Tensor.e21(i,j,k)*Tensor.e21(i,j,k) +
                        Tensor.e23(i,j,k)*Tensor.e23(i,j,k) +
                        Tensor.e31(i,j,k)*Tensor.e31(i,j,k) +
                        Tensor.e32(i,j,k)*Tensor.e32(i,j,k);
                       
     }
}
                         

template<typename Tprec>
__global__
void cuGetStrainTensor(TensorDev2<Tprec> Strain,
                       TensorDev2<Tprec> VelocityGradient
                      ){

     int i = blockDim.x*blockIdx.x + threadIdx.x;
     int j = blockDim.y*blockIdx.y + threadIdx.y;
     int k = blockDim.z*blockIdx.z + threadIdx.z;

     if(i < Strain.e11.getDim(1) && j < Strain.e11.getDim(2) 
                                 && k < Strain.e11.getDim(3)){

         Strain.e11(i,j,k) = 2.0*VelocityGradient.e11(i,j,k);
         Strain.e21(i,j,k) = (VelocityGradient.e21(i,j,k) + 
                              VelocityGradient.e12(i,j,k));
         Strain.e31(i,j,k) = (VelocityGradient.e31(i,j,k) + 
                              VelocityGradient.e13(i,j,k));

         Strain.e12(i,j,k) = (VelocityGradient.e12(i,j,k) + 
                              VelocityGradient.e21(i,j,k));
         Strain.e22(i,j,k) = 2.0*VelocityGradient.e22(i,j,k);
         Strain.e32(i,j,k) = (VelocityGradient.e32(i,j,k) + 
                              VelocityGradient.e23(i,j,k));


         Strain.e13(i,j,k) = (VelocityGradient.e13(i,j,k) + 
                              VelocityGradient.e31(i,j,k));
         Strain.e23(i,j,k) = (VelocityGradient.e23(i,j,k) + 
                              VelocityGradient.e32(i,j,k));
         Strain.e33(i,j,k) = 2.0*VelocityGradient.e33(i,j,k);


     }


}


template<typename Tprec>
__global__
void cuGetRotationTensor(TensorDev2<Tprec> Rotation,
                         TensorDev2<Tprec> VelocityGradient
                      ){

     int i = blockDim.x*blockIdx.x + threadIdx.x;
     int j = blockDim.y*blockIdx.y + threadIdx.y;
     int k = blockDim.z*blockIdx.z + threadIdx.z;

     if(i < Rotation.e11.getDim(1) && j < Rotation.e11.getDim(2) 
                                   && k < Rotation.e11.getDim(3)){

         Rotation.e11(i,j,k) = 0.0;
         Rotation.e21(i,j,k) = (VelocityGradient.e21(i,j,k) -
                                VelocityGradient.e12(i,j,k));
         Rotation.e31(i,j,k) = (VelocityGradient.e31(i,j,k) - 
                                VelocityGradient.e13(i,j,k));

         Rotation.e12(i,j,k) = (VelocityGradient.e12(i,j,k) -
                                VelocityGradient.e21(i,j,k));
         Rotation.e22(i,j,k) = 0.0;
         Rotation.e32(i,j,k) = (VelocityGradient.e32(i,j,k) - 
                                VelocityGradient.e23(i,j,k));


         Rotation.e13(i,j,k) = (VelocityGradient.e13(i,j,k) - 
                                VelocityGradient.e31(i,j,k));
         Rotation.e23(i,j,k) = (VelocityGradient.e23(i,j,k) - 
                                VelocityGradient.e32(i,j,k));
         Rotation.e33(i,j,k) = 0.0;


     }
}


template<typename Tprec>
__global__
void cuGetDyadicProduct(TensorDev2<Tprec> T2,
                        TensorDev2<Tprec> Tensor
                      ){

     int i = blockDim.x*blockIdx.x + threadIdx.x;
     int j = blockDim.y*blockIdx.y + threadIdx.y;
     int k = blockDim.z*blockIdx.z + threadIdx.z;

     if(i < T2.e11.getDim(1) && j < T2.e11.getDim(2) 
                                   && k < T2.e11.getDim(3)){
         // i=1, j=1
         T2.e11(i,j,k) = Tensor.e11(i,j,k)*Tensor.e11(i,j,k) +
                         Tensor.e12(i,j,k)*Tensor.e21(i,j,k) +
                         Tensor.e13(i,j,k)*Tensor.e31(i,j,k);

         // i=2, j=2
         T2.e22(i,j,k) = Tensor.e21(i,j,k)*Tensor.e12(i,j,k) +
                         Tensor.e22(i,j,k)*Tensor.e22(i,j,k) +
                         Tensor.e23(i,j,k)*Tensor.e32(i,j,k);

         // i=3, j=3
         T2.e33(i,j,k) = Tensor.e31(i,j,k)*Tensor.e13(i,j,k) +
                         Tensor.e32(i,j,k)*Tensor.e23(i,j,k) +
                         Tensor.e33(i,j,k)*Tensor.e33(i,j,k);

         // i=1, j=2
         T2.e12(i,j,k) = Tensor.e11(i,j,k)*Tensor.e12(i,j,k) +
                         Tensor.e12(i,j,k)*Tensor.e22(i,j,k) +
                         Tensor.e13(i,j,k)*Tensor.e32(i,j,k);

         // i=1, j=3
         T2.e13(i,j,k) = Tensor.e11(i,j,k)*Tensor.e13(i,j,k) +
                         Tensor.e12(i,j,k)*Tensor.e23(i,j,k) +
                         Tensor.e13(i,j,k)*Tensor.e33(i,j,k);

         // i=2, j=1
         T2.e21(i,j,k) = Tensor.e21(i,j,k)*Tensor.e11(i,j,k) +
                         Tensor.e22(i,j,k)*Tensor.e21(i,j,k) +
                         Tensor.e23(i,j,k)*Tensor.e31(i,j,k);

         // i=2, j=3
         T2.e23(i,j,k) = Tensor.e21(i,j,k)*Tensor.e13(i,j,k) +
                         Tensor.e22(i,j,k)*Tensor.e23(i,j,k) +
                         Tensor.e23(i,j,k)*Tensor.e33(i,j,k);

         // i=3, j=1
         T2.e31(i,j,k) = Tensor.e31(i,j,k)*Tensor.e11(i,j,k) +
                         Tensor.e32(i,j,k)*Tensor.e21(i,j,k) +
                         Tensor.e33(i,j,k)*Tensor.e31(i,j,k);

         // i=3, j=2
         T2.e32(i,j,k) = Tensor.e31(i,j,k)*Tensor.e12(i,j,k) +
                         Tensor.e32(i,j,k)*Tensor.e22(i,j,k) +
                         Tensor.e33(i,j,k)*Tensor.e32(i,j,k);
     }

}
///
                       
                        
template<typename Tprec>
void getStrainTensor(TensorDev2<Tprec>& Strain, 
                     TensorDev2<Tprec>& VelocityGradient
                    ){

    int gridDimX, gridDimY, gridDimZ;

    gridDimX = Strain.e11.getDim(1) / 8; 
    if(Strain.e11.getDim(1) % 8) gridDimX++;

    gridDimY = Strain.e11.getDim(2) / 8; 
    if(Strain.e11.getDim(2) % 8) gridDimY++;

    gridDimZ = Strain.e11.getDim(3) / 4; 
    if(Strain.e11.getDim(3) % 4) gridDimZ++;

    dim3 BlockDim(8,8,4);
    dim3 GridDim(gridDimX,gridDimY,gridDimZ);

    cuGetStrainTensor<<<GridDim, BlockDim>>>(Strain, VelocityGradient);

}

template<typename Tprec>
void getRotationTensor(TensorDev2<Tprec>& Rotation, 
                       TensorDev2<Tprec>& VelocityGradient
                    ){

    int gridDimX, gridDimY, gridDimZ;

    gridDimX = Rotation.e11.getDim(1) / 8; 
    if(Rotation.e11.getDim(1) % 8) gridDimX++;

    gridDimY = Rotation.e11.getDim(2) / 8; 
    if(Rotation.e11.getDim(2) % 8) gridDimY++;

    gridDimZ = Rotation.e11.getDim(3) / 4; 
    if(Rotation.e11.getDim(3) % 4) gridDimZ++;

    dim3 BlockDim(8,8,4);
    dim3 GridDim(gridDimX,gridDimY,gridDimZ);

    cuGetRotationTensor<<<GridDim, BlockDim>>>(Rotation, VelocityGradient);

}

template<typename Tprec>
void getDyadicProduct(TensorDev2<Tprec>& T2, 
                      TensorDev2<Tprec>& Tensor
                    ){

    int gridDimX, gridDimY, gridDimZ;

    gridDimX = Tensor.e11.getDim(1) / 8; 
    if(Tensor.e11.getDim(1) % 8) gridDimX++;

    gridDimY = Tensor.e11.getDim(2) / 8; 
    if(Tensor.e11.getDim(2) % 8) gridDimY++;

    gridDimZ = Tensor.e11.getDim(3) / 4; 
    if(Tensor.e11.getDim(3) % 4) gridDimZ++;

    dim3 BlockDim(8,8,4);
    dim3 GridDim(gridDimX,gridDimY,gridDimZ);

    cuGetStrainTensor<<<GridDim, BlockDim>>>(T2, Tensor);

}

template<typename Tprec>
void getDoubleContraction(ArrayDev3D<Tprec>& result, 
                          TensorDev2<Tprec>& Tensor
                         ){

    int gridDimX, gridDimY, gridDimZ;

    gridDimX = Tensor.e11.getDim(1) / 8; 
    if(Tensor.e11.getDim(1) % 8) gridDimX++;

    gridDimY = Tensor.e11.getDim(2) / 8; 
    if(Tensor.e11.getDim(2) % 8) gridDimY++;

    gridDimZ = Tensor.e11.getDim(3) / 4; 
    if(Tensor.e11.getDim(3) % 4) gridDimZ++;

    dim3 BlockDim(8,8,4);
    dim3 GridDim(gridDimX,gridDimY,gridDimZ);

    cuGetDoubleContraction<<<GridDim, BlockDim>>>(result, Tensor);

}

template<typename Tprec>
void getSdTensorWALLE(TensorDev2<Tprec>& Sd,
                      TensorDev2<Tprec>& S2,
                      TensorDev2<Tprec>& R2,
                      ArrayDev3D<Tprec>& SS, 
                      ArrayDev3D<Tprec>& RR
                     ){
    
    int gridDimX, gridDimY, gridDimZ;

    gridDimX = Sd.e11.getDim(1) / 8; 
    if(Sd.e11.getDim(1) % 8) gridDimX++;

    gridDimY = Sd.e11.getDim(2) / 8; 
    if(Sd.e11.getDim(2) % 8) gridDimY++;

    gridDimZ = Sd.e11.getDim(3) / 4; 
    if(Sd.e11.getDim(3) % 4) gridDimZ++;

    dim3 BlockDim(8,8,4);
    dim3 GridDim(gridDimX,gridDimY,gridDimZ);

    cuGetSdTensorWALLE<<<GridDim, BlockDim>>>(Sd, S2, R2, SS, RR);

}

template<typename Tprec>
void getViscousSGWALLE(ArrayDev3D<Tprec>& vt,
                       ArrayDev3D<Tprec>& Delta,
                       ArrayDev3D<Tprec>& SdSd,
                       ArrayDev3D<Tprec>& SS
                      ){

    int gridDimX, gridDimY, gridDimZ;

    gridDimX = SdSd.getDim(1) / 8; 
    if(SdSd.getDim(1) % 8) gridDimX++;

    gridDimY = SdSd.getDim(2) / 8; 
    if(SdSd.getDim(2) % 8) gridDimY++;

    gridDimZ = SdSd.getDim(3) / 4; 
    if(SdSd.getDim(3) % 4) gridDimZ++;

    dim3 BlockDim(8,8,4);
    dim3 GridDim(gridDimX,gridDimY,gridDimZ);

    cuGetViscousSGWALLE<<<GridDim, BlockDim>>>(vt, Delta, SdSd, SS);
     
}

template<typename Tprec>
void getGradientJacob(TensorDev2<Tprec>& Tensor,
                      ArrayDev3D<Tprec>& jbnX,
                      ArrayDev3D<Tprec>& jbnY,
                      ArrayDev3D<Tprec>& jbnZ
                      ){

    int gridDimX, gridDimY, gridDimZ;

    gridDimX = Tensor.e11.getDim(1) / 8; 
    if(Tensor.e11.getDim(1) % 8) gridDimX++;

    gridDimY = Tensor.e11.getDim(2) / 8; 
    if(Tensor.e11.getDim(2) % 8) gridDimY++;

    gridDimZ = Tensor.e11.getDim(3) / 4; 
    if(Tensor.e11.getDim(3) % 4) gridDimZ++;

    dim3 BlockDim(8,8,4);
    dim3 GridDim(gridDimX,gridDimY,gridDimZ);

    cuGetGradientJacob<<<GridDim, BlockDim>>>(Tensor, jbnX, jbnY, jbnZ);

}

extern "C"{

void cudaDeriv(
               double *dVel,
               double *dTemp,
               double *vel,
               double *temp,
               double *amut,
               double *dxyzsgd,
               double *jbn,
               double deltaX, double deltaY, double deltaZ,
               int Nx, int Ny, int Nz
              )
{

     int offset = Nx*Ny*Nz;

     //Cinematic variables on GPU device (Velocity)
     ArrayDev3D<double> u(Nx,Ny,Nz), v(Nx,Ny,Nz), w(Nx,Ny,Nz);

     //Thermodinamic variable on GPU device (Temperture)
     ArrayDev3D<double> T(Nx,Ny,Nz);

     //Gradient by element on GPU device (Velocity)
     ArrayDev3D<double> dudx(Nx,Ny,Nz), dudy(Nx,Ny,Nz), dudz(Nx,Ny,Nz);
     ArrayDev3D<double> dvdx(Nx,Ny,Nz), dvdy(Nx,Ny,Nz), dvdz(Nx,Ny,Nz);
     ArrayDev3D<double> dwdx(Nx,Ny,Nz), dwdy(Nx,Ny,Nz), dwdz(Nx,Ny,Nz);

     //Gradient by element on GPU device (Temperature and Pressure)
     ArrayDev3D<double> dTdx(Nx,Ny,Nz), dTdy(Nx,Ny,Nz), dTdz(Nx,Ny,Nz);
 //    ArrayDev3D<double> sinA(Nx,Ny,Nz), cosA(Nx,Ny,Nz);

     //std::cout<<offset << std::endl; 
/*
     for(int k = 0; k < Nz; k++){
        for(int j = 0; j < Ny; j++){
           for(int i = 0; i < Nx; i++){
              int id = i + j*Nx + k*Nx*Ny;
              temp[id] = sin(2.0*3.1415926*(double)k/((double)(Nz-1)));
           } 
        }
     }

      
     sinA.copyFromHost(temp);

     cu::cs::dFdZ(cosA, sinA, (2.0*3.1415926/((double)(Nz-1))));

     cosA.copyToHost(temp);    
 
     for(int k = 0; k < Nz; k++){
              int j = 1, i = 1;
              int id = i + j*Nx + k*Nx*Ny;
              std::cout<< temp[id]
              <<" "<< temp[id] << std::endl;
     } 

*/
/*
     //Copy from host
*/
     u.copyFromHost(&(vel[0*offset]));
     v.copyFromHost(&(vel[1*offset]));
     w.copyFromHost(&(vel[2*offset]));

     T.copyFromHost(temp);
     
/*
     //Gradients
*/

    //Velocity
      cu::cs::dFdX(dudx, u, deltaX);
      cu::cs::dFdY(dudy, u, deltaY);
      cu::cs::dFdZ(dudz, u, deltaZ);

       cu::cs::dFdX(dvdx, v, deltaX);
      cu::cs::dFdY(dvdy, v, deltaY);
      cu::cs::dFdZ(dvdz, v, deltaZ);

      cu::cs::dFdX(dwdx, w, deltaX);
      cu::cs::dFdY(dwdy, w, deltaY);
      cu::cs::dFdZ(dwdz, w, deltaZ);

     //Temperute
      cu::cs::dFdX(dTdx, T, deltaX);
      cu::cs::dFdY(dTdy, T, deltaY);
      cu::cs::dFdZ(dTdz, T, deltaZ);

     hipError_t error = hipGetLastError();
     if(error != hipSuccess)
     {
        // print the CUDA error message and exit
        printf("CUDA Derivadas error: %s\n", hipGetErrorString(error));
        exit(-1);
     }


/*
     //Copy values to Host
*/
     //Velocity Gradient without jacobian
     dudx.copyToHost(&(dVel[0*offset]));
     dvdx.copyToHost(&(dVel[1*offset]));
     dwdx.copyToHost(&(dVel[2*offset]));

     dudy.copyToHost(&(dVel[3*offset]));
     dvdy.copyToHost(&(dVel[4*offset]));
     dwdy.copyToHost(&(dVel[5*offset]));

     dudz.copyToHost(&(dVel[6*offset]));
     dvdz.copyToHost(&(dVel[7*offset]));
     dwdz.copyToHost(&(dVel[8*offset]));

     //Temperature Gradient without jacobian
     dTdx.copyToHost(&(dTemp[0*offset]));
     dTdy.copyToHost(&(dTemp[1*offset]));
     dTdz.copyToHost(&(dTemp[2*offset]));

     error = hipGetLastError();
     if(error != hipSuccess)
     {
        // print the CUDA error message and exit
        printf("CUDA Todo error: %s\n", hipGetErrorString(error));
        exit(-1);
     }

}

}
